#include "hip/hip_runtime.h"
/**
See the topology https://github.com/research-team/memristive-spinal-cord/blob/master/doc/diagram/cpg_generator_FE_paper.png
Based on the NEURON repository.
*/
#include <random>
#include <vector>
#include <string>
#include "test.h"
#include <stdexcept>
// for file writing
#include <cstdlib>
#include <iostream>
#include <fstream>
#include <unistd.h>
#define HANDLE_ERROR( err ) ( HandleError( err, __FILE__, __LINE__ ) )
#define PI 3.141592654f

using namespace std;

random_device r;
default_random_engine rand_gen(r());

static void HandleError(hipError_t err, const char *file, int line) {
	if (err != hipSuccess) {
		printf("!!! %s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit(EXIT_FAILURE);
	}
}

const double dt = 0.025;      // [ms] simulation step
const bool EXTRACELLULAR = false;

// global name of the models
const char GENERATOR = 'g';
const char INTER = 'i';
const char MOTO = 'm';
const char MUSCLE = 'u';
const char AFFERENTS = 'a';

const char layers = 5;      // number of OM layers (5 is default)
const int skin_time = 25;   // duration of layer 25 = 21 cm/s; 50 = 15 cm/s; 125 = 6 cm/s
const int step_number = 1;  // [step] number of full cycle steps
const int cv_fr = 200;      // frequency of CV
const int ees_fr = 40;      // frequency of EES
const int flexor_dur = 125; // flexor duration (125 or 175 ms for 4pedal)

const unsigned int one_step_time = 6 * skin_time + 125;
const unsigned int sim_time = 25 + one_step_time * step_number;
const auto SIM_TIME_IN_STEPS = (unsigned int)(sim_time / dt);  // [steps] converted time into steps

unsigned int nrns_number = 0;     // [id] global neuron id = number of neurons
unsigned int nrns_and_segs = 0;   // [id] global neuron+segs id = number of neurons with segments
const int neurons_in_group = 50;  // number of neurons in a group
const int neurons_in_ip = 196;    // number of neurons in a group

// common neuron constants
const double k = 0.01;            // synaptic coef
const double V_th = -40;          // [mV] voltage threshold
const double V_adj = -63;         // [mV] adjust voltage for -55 threshold
// moto neuron constants
const double ca0 = 2;             // initial calcium concentration
const double amA = 0.4;           // const ??? todo
const double amB = 66;            // const ??? todo
const double amC = 5;             // const ??? todo
const double bmA = 0.4;           // const ??? todo
const double bmB = 32;            // const ??? todo
const double bmC = 5;             // const ??? todo
const double R_const = 8.314472;  // [k-mole] or [joule/degC] const
const double F_const = 96485.34;  // [faraday] or [kilocoulombs] const
// muscle fiber constants
// const double g_kno = 0.01;     // [S/cm2] conductance of the todo
// const double g_kir = 0.03;     // [S/cm2] conductance of the Inwardly Rectifying Potassium K+ (Kir) channel
// Boltzman steady state curve
const double vhalfl = -98.92;     // [mV] inactivation half-potential
const double kl = 10.89;          // [mV] Stegen et al. 2012
// tau_infty
const double vhalft = 67.0828;    // [mV] fitted //100 uM sens curr 350a, Stegen et al. 2012
const double at = 0.00610779;     // [/ ms] Stegen et al. 2012
const double bt = 0.0817741;      // [/ ms] Note: typo in Stegen et al. 2012
// temperature dependence
const double q10 = 1;             // temperature scaling (sensitivity)
const double celsius = 36;        // [degC] temperature of the cell
// i_membrane [mA/cm2]
//const double e_extracellular = 0; // [mV]
//const double xraxial = 1e9;       // [MOhm/cm]

// neuron parameters
vector<unsigned int> vector_nrn_start_seg;
vector<char> vector_models;
vector<double> vector_Cm, vector_gnabar, vector_gkbar, vector_gl, vector_Ra, vector_diam, vector_length, vector_ena,
               vector_ek, vector_el, vector_gkrect, vector_gcaN, vector_gcaL, vector_gcak;
// synaptic parameters
vector<double> vector_E_ex, vector_E_inh, vector_tau_exc, vector_tau_inh1, vector_tau_inh2;
// synapses varaibels
vector<int> vector_syn_pre_nrn, vector_syn_post_nrn, vector_syn_delay, vector_syn_delay_timer;
vector<double> vector_syn_weight;
// results vector
vector <GroupMetadata> saving_groups;
// for debugging
vector <Group> all_groups;
// generators
vector<unsigned int> vec_time_end, vec_nrn_id, vec_freq_in_steps, vec_spike_each_step;

// form structs of neurons global ID and groups name
Group form_group(const string &group_name,
	             int nrns_in_group = neurons_in_group,
	             const char model = INTER,
	             const int segs = 1) {
	/**
	 *
	 */
	Group group = Group();
	group.group_name = group_name;     // name of a neurons group
	group.id_start = nrns_number;      // first ID in the group
	group.id_end = nrns_number + nrns_in_group - 1;  // the latest ID in the group
	group.group_size = nrns_in_group;  // size of the neurons group

	double Cm, gnabar, gkbar, gl, Ra, ena, ek, el, diam, dx, gkrect, gcaN, gcaL, gcak, e_ex, e_inh, tau_exc, tau_inh1, tau_inh2;
	normal_distribution<double> Cm_distr(1, 0.01);
	uniform_int_distribution<int> moto_diam_distr(45, 55);
	uniform_int_distribution<int> inter_diam_distr(5, 15);
	uniform_real_distribution<double> afferent_diam_distr(15, 35);

	for (int nrn = 0; nrn < nrns_in_group; nrn++) {
		if (model == INTER) {
			Cm = Cm_distr(rand_gen);
			gnabar = 0.1;
			gkbar = 0.08;
			gl = 0.002;
			Ra = 100.0;
			ena = 50.0;
			ek = -90.0;
			el = -70.0;
			diam = inter_diam_distr(rand_gen); // 10
			dx = diam;
			e_ex = 50;
			e_inh = -80;
			tau_exc = 0.35;
			tau_inh1 = 0.5;
			tau_inh2 = 3.5;
		} else if (model == AFFERENTS) {
			Cm = 2;
			gnabar = 0.5;
			gkbar = 0.04;
			gl = 0.002;
			Ra = 200.0;
			ena = 50.0;
			ek = -90.0;
			el = -70.0;
			diam = afferent_diam_distr(rand_gen); // 10
			dx = diam;
			e_ex = 50;
			e_inh = -80;
			tau_exc = 0.35;
			tau_inh1 = 0.5;
			tau_inh2 = 3.5;
		} else if (model == MOTO) {
			Cm = 2;
			gnabar = 0.05;
			gl = 0.002;
			Ra = 200.0;
			ena = 50.0;
			ek = -80.0;
			el = -70.0;
			diam = moto_diam_distr(rand_gen);
			dx = diam;
			gkrect = 0.3;
			gcaN = 0.05;
			gcaL = 0.0001;
			gcak = 0.3;
			e_ex = 50.0;
			e_inh = -80.0;
			tau_exc = 0.3;
			tau_inh1 = 1.0;
			tau_inh2 = 1.5;
			if (diam > 50) {
				gnabar = 0.1;
				gcaL = 0.001;
				gl = 0.003;
				gkrect = 0.2;
				gcak = 0.2;
			}
		} else if (model == MUSCLE) {
			Cm = 3.6;
			gnabar = 0.15;
			gkbar = 0.03;
			gl = 0.0002;
			Ra = 1.1;
			ena = 55.0;
			ek = -80.0;
			el = -72.0;
			diam = 40.0;
			dx = 3000.0;
			e_ex = 0.0;
			e_inh = -80.0;
			tau_exc = 0.3;
			tau_inh1 = 1.0;
			tau_inh2 = 1.0;
		} else if (model == GENERATOR) {

		} else {
			throw logic_error("Choose the model");
		}
		// common properties
		vector_Cm.push_back(Cm);
		vector_gnabar.push_back(gnabar);
		vector_gkbar.push_back(gkbar);
		vector_gl.push_back(gl);
		vector_el.push_back(el);
		vector_ena.push_back(ena);
		vector_ek.push_back(ek);
		vector_Ra.push_back(Ra);
		vector_diam.push_back(diam);
		vector_length.push_back(dx);
		vector_gkrect.push_back(gkrect);
		vector_gcaN.push_back(gcaN);
		vector_gcaL.push_back(gcaL);
		vector_gcak.push_back(gcak);
		vector_E_ex.push_back(e_ex);
		vector_E_inh.push_back(e_inh);
		vector_tau_exc.push_back(tau_exc);
		vector_tau_inh1.push_back(tau_inh1);
		vector_tau_inh2.push_back(tau_inh2);
		//
		vector_nrn_start_seg.push_back(nrns_and_segs);
		nrns_and_segs += (segs + 2);
		vector_models.push_back(model);
	}

	nrns_number += nrns_in_group;
	printf("Formed %s IDs [%d ... %d] = %d\n",
	       group_name.c_str(), nrns_number - nrns_in_group, nrns_number - 1, nrns_in_group);

	// for debugging
	all_groups.push_back(group);

	return group;
}

__host__
unsigned int ms_to_step(double ms) { return (unsigned int) (ms / dt); }

__host__
double step_to_ms(int step) { return step * dt; }

// copy data from host to device
template<typename type>
void memcpyHtD(type *host, type *gpu, unsigned int size) {
	HANDLE_ERROR(hipMemcpy(gpu, host, sizeof(type) * size, hipMemcpyHostToDevice));
}

// copy data from device to host
template<typename type>
void memcpyDtH(type *gpu, type *host, unsigned int size) {
	HANDLE_ERROR(hipMemcpy(host, gpu, size * sizeof(type), hipMemcpyDeviceToHost));
}

// init GPU array and copy data from the CPU array
template<typename type>
type* init_gpu_arr(type *cpu_var, unsigned int size = nrns_and_segs) {
	type *gpu_var;
	HANDLE_ERROR(hipMalloc(&gpu_var, size * sizeof(type)));
	memcpyHtD<type>(cpu_var, gpu_var, size);
	return gpu_var;
}

// init GPU array and copy data from the CPU vector
template<typename type>
type *init_gpu_arr(vector<type> &vec) {
	type *gpu_var;
	HANDLE_ERROR(hipMalloc(&gpu_var, sizeof(type) * vec.size()));
	memcpyHtD<type>(vec.data(), gpu_var, vec.size());
	return gpu_var;
}

void add_generator(Group &group, double start, double end, double freq) {
	vec_nrn_id.push_back(group.id_start);
	vec_time_end.push_back(ms_to_step(end));
	vec_freq_in_steps.push_back(ms_to_step(1000 / freq));
	vec_spike_each_step.push_back(ms_to_step(start));
	printf("start %d end %d freq %d\n", ms_to_step(start), ms_to_step(end), ms_to_step(1000 / freq));
}

// convert vector to the array
template<typename type>
type* vec2arr(vector<type> &vec) {
	return vec.cpu_vector.data();
}

__device__
double Exp(double volt) {
	return (volt < -100)? 0 : exp(volt);
}

__device__
double alpham(double volt) {
	if (abs((volt + amB) / amC) < 1e-6)
		return amA * amC;
	return amA * (volt + amB) / (1.0 - Exp(-(volt + amB) / amC));
}

__device__
double betam(double volt) {
	if (abs((volt + bmB) / bmC) < 1e-6)
		return -bmA * bmC;
	return -bmA * (volt + bmB) / (1.0 - Exp((volt + bmB) / bmC));
}

__device__
double syn_current(Neurons* N, Parameters* P, int nrn, double voltage) {
	/**
	 * calculate synaptic current
	 */
	return N->g_exc[nrn] * (voltage - P->E_ex[nrn]) + (N->g_inh_B[nrn] - N->g_inh_A[nrn]) * (voltage - P->E_inh[nrn]);
}

__device__
double nrn_moto_current(States* S, Parameters* P, Neurons* N, int nrn, int nrn_seg_index, double voltage) {
	/**
	 * calculate channels current
	 */
	double iNa = P->gnabar[nrn] * pow(S->m[nrn_seg_index], 3) * S->h[nrn_seg_index] * (voltage - P->ena[nrn]);
	double iK = P->gkrect[nrn] * pow(S->n[nrn_seg_index], 4) * (voltage - P->ek[nrn]) +
                P->gcak[nrn] * pow(S->cai[nrn_seg_index], 2) / (pow(S->cai[nrn_seg_index], 2) + 0.014 * 0.014) * (voltage - P->ek[nrn]);
	double iL = P->gl[nrn] * (voltage - P->el[nrn]);
	double eCa = (1000 * R_const * 309.15 / (2 * F_const)) * log(ca0 / S->cai[nrn_seg_index]);
	S->I_Ca[nrn_seg_index] = P->gcaN[nrn] * pow(S->mc[nrn_seg_index], 2) * S->hc[nrn_seg_index] * (voltage - eCa) +
	                         P->gcaL[nrn] * S->p[nrn_seg_index] * (voltage - eCa);
	return iNa + iK + iL + S->I_Ca[nrn_seg_index];
}

__device__
double nrn_fastchannel_current(States* S, Parameters* P, Neurons* N, int nrn, int nrn_seg_index, double voltage) {
	/**
	 * calculate channels current
	 */
	double iNa = P->gnabar[nrn] * pow(S->m[nrn_seg_index], 3) * S->h[nrn_seg_index] * (voltage - P->ena[nrn]);
	double iK = P->gkbar[nrn] * pow(S->n[nrn_seg_index], 4) * (voltage - P->ek[nrn]);
	double iL = P->gl[nrn] * (voltage - P->el[nrn]);
	return iNa + iK + iL;
}

__device__
void recalc_synaptic(States* S, Parameters* P, Neurons* N, int nrn) {
	/**
	 * updating conductance(summed) of neurons' post-synaptic conenctions
	 */
	// exc synaptic conductance
	if (N->g_exc[nrn] != 0) {
		N->g_exc[nrn] -= (1.0 - exp(-dt / P->tau_exc[nrn])) * N->g_exc[nrn];
		if (N->g_exc[nrn] < 1e-5) {
			N->g_exc[nrn] = 0.0;
		}
	}
	// inh1 synaptic conductance
	if (N->g_inh_A[nrn] != 0) {
		N->g_inh_A[nrn] -= (1.0 - exp(-dt / P->tau_inh1[nrn])) * N->g_inh_A[nrn];
		if (N->g_inh_A[nrn] < 1e-5) {
			N->g_inh_A[nrn] = 0.0;
		}
	}
	// inh2 synaptic conductance
	if (N->g_inh_B[nrn] != 0) {
		N->g_inh_B[nrn] -= (1.0 - exp(-dt / P->tau_inh2[nrn])) * N->g_inh_B[nrn];
		if (N->g_inh_B[nrn] < 1e-5)
			N->g_inh_B[nrn] = 0.0;
	}
}

__device__
void syn_initial(States* S, Parameters* P, Neurons* N, int nrn) {
	/**
	 * initialize tau(rise / decay time, ms) and factor(const) variables
	 */
	if (P->tau_inh1[nrn] / P->tau_inh2[nrn] > 0.9999)
		P->tau_inh1[nrn] = 0.9999 * P->tau_inh2[nrn];
	if (P->tau_inh1[nrn] / P->tau_inh2[nrn] < 1e-9)
		P->tau_inh1[nrn] = P->tau_inh2[nrn] * 1e-9;
	//
	double tp = (P->tau_inh1[nrn] * P->tau_inh2[nrn]) / (P->tau_inh2[nrn] - P->tau_inh1[nrn]) *
	           log(P->tau_inh2[nrn] / P->tau_inh1[nrn]);
	N->factor[nrn] = -exp(-tp / P->tau_inh1[nrn]) + exp(-tp / P->tau_inh2[nrn]);
	N->factor[nrn] = 1.0 / N->factor[nrn];
}

__device__
void nrn_inter_initial(States* S, Parameters* P, Neurons* N, int nrn_seg_index, double V) {
	/**
	 * initialize channels, based on cropped evaluate_fct function
	 */
	double V_mem = V - V_adj;
	//
	double a = 0.32 * (13.0 - V_mem) / (exp((13.0 - V_mem) / 4.0) - 1.0);
	double b = 0.28 * (V_mem - 40.0) / (exp((V_mem - 40.0) / 5.0) - 1.0);
	S->m[nrn_seg_index] = a / (a + b);   // m_inf
	//
	a = 0.128 * exp((17.0 - V_mem) / 18.0);
	b = 4.0 / (1.0 + exp((40.0 - V_mem) / 5.0));
	S->h[nrn_seg_index] = a / (a + b);   // h_inf
	//
	a = 0.032 * (15.0 - V_mem) / (exp((15.0 - V_mem) / 5.0) - 1.0);
	b = 0.5 * exp((10.0 - V_mem) / 40.0);
	S->n[nrn_seg_index] = a / (a + b);   // n_inf
}

__device__
void nrn_moto_initial(States* S, Parameters* P, Neurons* N, int nrn_seg_index, double V) {
	/**
	 * initialize channels, based on cropped evaluate_fct function
	 */
	double a = alpham(V);
	S->m[nrn_seg_index] = a / (a + betam(V));                         // m_inf
	S->h[nrn_seg_index] = 1.0 / (1.0 + Exp((V + 65.0) / 7.0));   // h_inf
	S->p[nrn_seg_index] = 1.0 / (1.0 + Exp(-(V + 55.8) / 3.7));  // p_inf
	S->n[nrn_seg_index] = 1.0 / (1.0 + Exp(-(V + 38.0) / 15.0)); // n_inf
	S->mc[nrn_seg_index] = 1.0 / (1.0 + Exp(-(V + 32.0) / 5.0)); // mc_inf
	S->hc[nrn_seg_index] = 1.0 / (1.0 + Exp((V + 50.0) / 5.0));  // hc_inf
	S->cai[nrn_seg_index] = 0.0001;
}

__device__
void nrn_muslce_initial(States* S, Parameters* P, Neurons* N, int nrn_seg_index, double V) {
	/**
	 * initialize channels, based on cropped evaluate_fct function
	 */
	double V_mem = V - V_adj;
	// m_inf
	double a = 0.32 * (13.0 - V_mem) / (exp((13.0 - V_mem) / 4.0) - 1.0);
	double b = 0.28 * (V_mem - 40.0) / (exp((V_mem - 40.0) / 5.0) - 1.0);
	S->m[nrn_seg_index] = a / (a + b);
	// h_inf
	a = 0.128 * exp((17.0 - V_mem) / 18.0);
	b = 4.0 / (1.0 + exp((40.0 - V_mem) / 5.0));
	S->h[nrn_seg_index] = a / (a + b);
	// n_inf
	a = 0.032 * (15.0 - V_mem) / (exp((15.0 - V_mem) / 5.0) - 1.0);
	b = 0.5 * exp((10.0 - V_mem) / 40.0);
	S->n[nrn_seg_index] = a / (a + b);
}

__device__
void recalc_inter_channels(States* S, Parameters* P, Neurons* N, int nrn_seg_index, double V) {
	/**
	 * calculate new states of channels (evaluate_fct)
	 */
	// BREAKPOINT -> states -> evaluate_fct
	double V_mem = V - V_adj;
	//
	double a = 0.32 * (13.0 - V_mem) / (exp((13.0 - V_mem) / 4.0) - 1.0);
	double b = 0.28 * (V_mem - 40.0) / (exp((V_mem - 40.0) / 5.0) - 1.0);
	double tau = 1.0 / (a + b);
	double inf = a / (a + b);
	S->m[nrn_seg_index] += (1.0 - exp(-dt / tau)) * (inf - S->m[nrn_seg_index]);
	//
	a = 0.128 * exp((17.0 - V_mem) / 18.0);
	b = 4.0 / (1.0 + exp((40.0 - V_mem) / 5.0));
	tau = 1.0 / (a + b);
	inf = a / (a + b);
	S->h[nrn_seg_index] += (1.0 - exp(-dt / tau)) * (inf - S->h[nrn_seg_index]);
	//
	a = 0.032 * (15.0 - V_mem) / (exp((15.0 - V_mem) / 5.0) - 1.0);
	b = 0.5 * exp((10.0 - V_mem) / 40.0);
	tau = 1.0 / (a + b);
	inf = a / (a + b);
	// states
	S->n[nrn_seg_index] += (1.0 - exp(-dt / tau)) * (inf - S->n[nrn_seg_index]);
}

__device__
void recalc_moto_channels(States* S, Parameters* P, Neurons* N, int nrn_seg_index, double V) {
	/**
	 * calculate new states of channels (evaluate_fct)
	 */
	// BREAKPOINT -> states -> evaluate_fct
	double a = alpham(V);
	double b = betam(V);
	// m
	double tau = 1.0 / (a + b);
	double inf = a / (a + b);
	S->m[nrn_seg_index] += (1.0 - exp(-dt / tau)) * (inf - S->m[nrn_seg_index]);
	// h
	tau = 30.0 / (Exp((V + 60.0) / 15.0) + Exp(-(V + 60.0) / 16.0));
	inf = 1.0 / (1 + Exp((V + 65.0) / 7.0));
	S->h[nrn_seg_index] += (1.0 - exp(-dt / tau)) * (inf - S->h[nrn_seg_index]);
	// DELAYED RECTIFIER POTASSIUM
	tau = 5.0 / (Exp((V + 50.0) / 40.0) + Exp(-(V + 50.0) / 50.0));
	inf = 1.0 / (1.0 + Exp(-(V + 38.0) / 15.0));
	S->n[nrn_seg_index] += (1.0 - exp(-dt / tau)) * (inf - S->n[nrn_seg_index]);
	// CALCIUM DYNAMICS N-type
	double mc_inf = 1.0 / (1.0 + Exp(-(V + 32.0) / 5.0));
	double hc_inf = 1.0 / (1.0 + Exp((V + 50.0) / 5.0));
	// CALCIUM DYNAMICS L-type
	tau = 400.0;
	inf = 1.0 / (1.0 + Exp(-(V + 55.8) / 3.7));
	S->p[nrn_seg_index] += (1.0 - exp(-dt / tau)) * (inf - S->p[nrn_seg_index]);
	// states
	S->mc[nrn_seg_index] += (1.0 - exp(-dt / 15.0)) * (mc_inf - S->mc[nrn_seg_index]);     // tau_mc = 15
	S->hc[nrn_seg_index] += (1.0 - exp(-dt / 50.0)) * (hc_inf - S->hc[nrn_seg_index]);     // tau_hc = 50
	S->cai[nrn_seg_index] += (1.0 - exp(-dt * 0.04)) * (-0.01 * S->I_Ca[nrn_seg_index] / 0.04 - S->cai[nrn_seg_index]);
}

__device__
void recalc_muslce_channels(States* S, Parameters* P, Neurons* N, int nrn_seg_index, double V) {
	/**
	 * calculate new states of channels (evaluate_fct)
	 */
	// BREAKPOINT -> states -> evaluate_fct
	double V_mem = V - V_adj;
	//
	double a = 0.32 * (13.0 - V_mem) / (exp((13.0 - V_mem) / 4.0) - 1.0);
	double b = 0.28 * (V_mem - 40.0) / (exp((V_mem - 40.0) / 5.0) - 1.0);
	double tau = 1.0 / (a + b);
	double inf = a / (a + b);
	S->m[nrn_seg_index] += (1.0 - exp(-dt / tau)) * (inf - S->m[nrn_seg_index]);
	//
	a = 0.128 * exp((17.0 - V_mem) / 18.0);
	b = 4.0 / (1.0 + exp((40.0 - V_mem) / 5.0));
	tau = 1.0 / (a + b);
	inf = a / (a + b);
	S->h[nrn_seg_index] += (1.0 - exp(-dt / tau)) * (inf - S->h[nrn_seg_index]);
	//
	a = 0.032 * (15.0 - V_mem) / (exp((15.0 - V_mem) / 5.0) - 1.0);
	b = 0.5 * exp((10.0 - V_mem) / 40.0);
	tau = 1.0 / (a + b);
	inf = a / (a + b);
	S->n[nrn_seg_index] += (1.0 - exp(-dt / tau)) * (inf - S->n[nrn_seg_index]);
	//
	double qt = pow(q10, (celsius - 33.0) / 10.0);
	double linf = 1.0 / (1.0 + exp((V - vhalfl) / kl)); // l_steadystate
	double taul = 1.0 / (qt * (at * exp(-V / vhalft) + bt * exp(V / vhalft)));
	double alpha = 0.3 / (1.0 + exp((V + 43.0) / -5.0));
	double beta = 0.03 / (1.0 + exp((V + 80.0) / -1.0));
	double stau = 1.0 / (alpha + beta);
	double sinf = alpha / (alpha + beta);
	// states
	S->l[nrn_seg_index] += (1.0 - exp(-dt / taul)) * (linf - S->l[nrn_seg_index]);
	S->s[nrn_seg_index] += (1.0 - exp(-dt / stau)) * (sinf - S->s[nrn_seg_index]);
}

__device__
void nrn_rhs_ext(int nrn) {

}

__device__
void nrn_setup_ext(int nrn) {

}

__device__
void nrn_update_2d(int nrn) {

}

__device__
void nrn_rhs(States* S, Parameters* P, Neurons* N, int nrn, int i1, int i3) {
	/**
	 * void nrn_rhs(NrnThread *_nt) combined with the first part of nrn_lhs
	 * calculate right hand side of
	 * cm*dvm/dt = -i(vm) + is(vi) + ai_j*(vi_j - vi)
	 * cx*dvx/dt - cm*dvm/dt = -gx*(vx - ex) + i(vm) + ax_j*(vx_j - vx)
	 * This is a common operation for fixed step, cvode, and daspk methods
	 */
	// init _rhs and _lhs (NODE_D) as zero
	for (int i = i1; i < i3; ++i) {
		S->NODE_RHS[i] = 0.0;
		S->NODE_D[i] = 0.0;
//		ext_rhs[i1:i3, :] = 0
	}

	// update MOD rhs, CAPS has no current [CAP MOD CAP]!
	int center_segment = i1 + ((P->models[nrn] == MUSCLE)? 2 : 1);
	// update segments except CAPs
	double V, _g, _rhs;
	for (int nrn_seg = i1 + 1; nrn_seg < i3 - 1; ++nrn_seg) {
		V = S->Vm[nrn_seg];
		// SYNAPTIC update
		if (nrn_seg == center_segment) {
			// static void nrn_cur(_NrnThread* _nt, _Memb_list* _ml, int _type)
			_g = syn_current(N, P, nrn, V + 0.001);
			_rhs = syn_current(N, P, nrn, V);
			_g = (_g - _rhs) / 0.001;
			_g *= 1.e2 / S->NODE_AREA[nrn_seg];
			_rhs *= 1.e2 / S->NODE_AREA[nrn_seg];
			S->NODE_RHS[nrn_seg] -= _rhs;
			// static void nrn_jacob(_NrnThread* _nt, _Memb_list* _ml, int _type)
			S->NODE_D[nrn_seg] += _g;
		}
		// NEURON update
		// static void nrn_cur(_NrnThread* _nt, _Memb_list* _ml, int _type)
		if (P->models[nrn] == INTER || P->models[nrn] == AFFERENTS) {
			// muscle and inter has the same fast_channel function
			_g = nrn_fastchannel_current(S, P, N, nrn, nrn_seg, V + 0.001);
			_rhs = nrn_fastchannel_current(S, P, N, nrn, nrn_seg, V);
		} else if (P->models[nrn] == MOTO) {
			_g = nrn_moto_current(S, P, N, nrn, nrn_seg, V + 0.001);
			_rhs = nrn_moto_current(S, P, N, nrn, nrn_seg, V);
		} else if (P->models[nrn] == MUSCLE) {
			// muscle and inter has the same fast_channel function
			_g = nrn_fastchannel_current(S, P, N, nrn, nrn_seg, V + 0.001);
			_rhs = nrn_fastchannel_current(S, P, N, nrn, nrn_seg, V);
		} else {
			// todo
		}
		// save data like in NEURON (after .mod nrn_cur)
		_g = (_g - _rhs) / 0.001;
		S->NODE_RHS[nrn_seg] -= _rhs;
		// static void nrn_jacob(_NrnThread* _nt, _Memb_list* _ml, int _type)
		S->NODE_D[nrn_seg] += _g;
	} // end FOR segments
	// activsynapse_rhs()
	if (EXTRACELLULAR) {
		// Cannot have any axial terms yet so that i(vm) can be calculated from
		// i(vm)+is(vi) and is(vi) which are stored in rhs vector.
		nrn_rhs_ext(nrn);
		// nrn_rhs_ext has also computed the the internal axial current for those
		// nodes containing the extracellular mechanism
	}
	// activstim_rhs()
	// activclamp_rhs()

	// todo: always 0, because Vm0 = Vm1 = Vm2 at [CAP node CAP] model (1 section)
	double dv;
	for (int nrn_seg = i1 + 1; nrn_seg < i3; ++nrn_seg) {
		dv = S->Vm[nrn_seg - 1] - S->Vm[nrn_seg];
		// our connection coefficients are negative so
		S->NODE_RHS[nrn_seg] -= S->NODE_B[nrn_seg] * dv;
		S->NODE_RHS[nrn_seg - 1] += S->NODE_A[nrn_seg] * dv;
	}
}

__device__
void bksub(States* S, Parameters* P, Neurons* N, int nrn, int i1, int i3) {
	/**
	 * void bksub(NrnThread* _nt)
	 */
	// intracellular
	S->NODE_RHS[i1] /= S->NODE_D[i1];
	//
	for (int nrn_seg = i1 + 1; nrn_seg < i3; ++nrn_seg) {
		S->NODE_RHS[nrn_seg] -= S->NODE_B[nrn_seg] * S->NODE_RHS[nrn_seg - 1];
		S->NODE_RHS[nrn_seg] /= S->NODE_D[nrn_seg];
	}
	// extracellular
	if (EXTRACELLULAR) {
	//	for j in range(nlayer):
	//	ext_rhs[i1, j] /= ext_d[i1, j]
	//	for nrn_seg in range(i1 + 1, i3):
	//	for j in range(nlayer):
	//	ext_rhs[nrn_seg, j] -= ext_b[nrn_seg, j] * ext_rhs[nrn_seg - 1, j]
	//	ext_rhs[nrn_seg, j] /= ext_d[nrn_seg, j]
	}
}

__device__
void triang(States* S, Parameters* P, Neurons* N, int nrn, int i1, int i3) {
	/**
	 * void triang(NrnThread* _nt)
	 */
	// intracellular
	double ppp;
	int nrn_seg = i3 - 1;
	while (nrn_seg >= i1 + 1) {
		ppp = S->NODE_A[nrn_seg] / S->NODE_D[nrn_seg];
		S->NODE_D[nrn_seg - 1] -= ppp * S->NODE_B[nrn_seg];
		S->NODE_RHS[nrn_seg - 1] -= ppp * S->NODE_RHS[nrn_seg];
		nrn_seg--;
	}
	// extracellular
	if (EXTRACELLULAR) {
//		nrn_seg = i3 - 1
//		while nrn_seg >= i1 + 1:
//			for j in range(nlayer):
//				ppp = ext_a[nrn_seg, j] / ext_d[nrn_seg, j]
//				ext_d[nrn_seg - 1, j] -= ppp * ext_b[nrn_seg, j]
//				ext_rhs[nrn_seg - 1, j] -= ppp * ext_rhs[nrn_seg, j]
//			nrn_seg--
	}
}

__device__
void nrn_solve(States* S, Parameters* P, Neurons* N, int nrn, int i1, int i3) {
	/**
	 * void nrn_solve(NrnThread* _nt)
	 */
	triang(S, P, N, nrn, i1, i3);
	bksub(S, P, N, nrn, i1, i3);
}

__device__
void setup_tree_matrix(States* S, Parameters* P, Neurons* N, int nrn, int i1, int i3) {
	/**
	 * void setup_tree_matrix(NrnThread* _nt)
	 */
	nrn_rhs(S, P, N, nrn, i1, i3);
	// simplified nrn_lhs(nrn)
	for (int i = i1; i < i3; ++i) {
		S->NODE_D[i] += S->const_NODE_D[i];
	}
}

__device__
void update(States* S, Parameters* P, Neurons* N, int nrn, int i1, int i3) {
	/**
	 * void update(NrnThread* _nt)
	 */
	// final voltage updating
	for (int nrn_seg = i1; nrn_seg < i3; ++nrn_seg) {
		S->Vm[nrn_seg] += S->NODE_RHS[nrn_seg];
	}
	// save data like in NEURON (after .mod nrn_cur)
//	if DEBUG and nrn in save_neuron_ids:
//	save_data()
	// extracellular
	nrn_update_2d(nrn);
}

__device__
void nrn_deliver_events(States* S, Parameters* P, Neurons* N, int nrn) {
	/**
	 * void nrn_deliver_events(NrnThread* nt)
	 */
	// get the central segment (for detecting spikes): i1 + (2 or 1)
	int seg_update = P->nrn_start_seg[nrn] + ((P->models[nrn] == MUSCLE)? 2 : 1);
	// check if neuron has spike with special flag for avoidance multi-spike detecting
	if (!N->spike_on[nrn] && S->Vm[seg_update] > V_th) {
		N->spike_on[nrn] = true;
		N->has_spike[nrn] = true;
	} else if (S->Vm[seg_update] < V_th) {
		N->spike_on[nrn] = false;
	}
}

__device__
void nrn_fixed_step_lastpart(States* S, Parameters* P, Neurons* N, int nrn, int i1, int i3) {
	/**
	 * void *nrn_fixed_step_lastpart(NrnThread *nth)
	 */
	// update neurons' synapses state
	recalc_synaptic(S, P, N, nrn);
	//  update neurons' segments state
	if (P->models[nrn] == INTER || P->models[nrn] == AFFERENTS) {
		for(int nrn_seg = i1; nrn_seg < i3; ++nrn_seg) {
			recalc_inter_channels(S, P, N, nrn_seg, S->Vm[nrn_seg]);
		}
	} else if (P->models[nrn] == MOTO) {
		for(int nrn_seg = i1; nrn_seg < i3; ++nrn_seg) {
			recalc_moto_channels(S, P, N, nrn_seg, S->Vm[nrn_seg]);
		}
	} else if (P->models[nrn] == MUSCLE) {
		for(int nrn_seg = i1; nrn_seg < i3; ++nrn_seg) {
			recalc_muslce_channels(S, P, N, nrn_seg, S->Vm[nrn_seg]);
		}
	} else {

	}
	//  spike detection for (in synapse kernel)
	nrn_deliver_events(S, P, N, nrn);
}

__device__
void nrn_area_ri(States* S, Parameters* P, Neurons* N) {
	/**
	 * void nrn_area_ri(Section *sec) [790] treeset.c
	 * area for right circular cylinders. Ri as right half of parent + left half of this
	 */
	printf("GPU: nrn_area_ri\n");
	double dx, rleft, rright;
	int i1, i3, nrn_seg, segments;
	//
	for (int nrn = 0; nrn < N->size; ++nrn) {
		if (P->models[nrn] == GENERATOR)
			continue;
		i1 = P->nrn_start_seg[nrn];
		i3 = P->nrn_start_seg[nrn + 1];
		segments = (i3 - i1 - 2);
		dx = P->length[nrn] / segments; // divide by the last index of node (or segments count)
		rright = 0;
		// todo sec->pnode needs +1 index
		for (nrn_seg = i1 + 1; nrn_seg < i1 + segments + 1; ++nrn_seg) {
			// area for right circular cylinders. Ri as right half of parent + left half of this
			S->NODE_AREA[nrn_seg] = PI * dx * P->diam[nrn];
			rleft = 1.e-2 * P->Ra[nrn] * (dx / 2.0) / (PI * pow(P->diam[nrn], 2) / 4.0);   // left half segment Megohms
			S->NODE_RINV[nrn_seg] = 1.0 / (rleft + rright); // uS
			rright = rleft;
		}
		//the first and last segments has zero length. Area is 1e2 in dimensionless units
		S->NODE_AREA[i1] = 100.0;
		nrn_seg = i1 + segments + 1; // the last segment
		S->NODE_AREA[nrn_seg] = 100.0;
		S->NODE_RINV[nrn_seg] = 1.0 / rright;
	}
}

__device__
void ext_con_coef(States* S, Parameters* P, Neurons* N) {

}

__device__
void connection_coef(States* S, Parameters* P, Neurons* N) {
	/**
	 * void connection_coef(void) treeset.c
	 */
	printf("GPU: connection_coef\n");
	nrn_area_ri(S, P, N);
	// NODE_A is the effect of this node on the parent node's equation
	// NODE_B is the effect of the parent node on this node's equation
	int i1, i3, nrn_seg, segments;
	//
	for (int nrn = 0; nrn < N->size; ++nrn) {
		if (P->models[nrn] == GENERATOR)
			continue;
		i1 = P->nrn_start_seg[nrn];
		i3 = P->nrn_start_seg[nrn + 1];
		segments = (i3 - i1 - 2);
		// first the effect of node on parent equation. Note that last nodes have area = 1.e2 in dimensionless
		// units so that last nodes have units of microsiemens
		// todo sec->pnode needs +1 index
		nrn_seg = i1 + 1;
		// sec->prop->dparam[4].val = 1, what is dparam[4].val
		S->NODE_A[nrn_seg] = -1.e2 * 1.0 * S->NODE_RINV[nrn_seg] / S->NODE_AREA[nrn_seg - 1];
		// todo sec->pnode needs +1 index
		for (nrn_seg = i1 + 1 + 1; nrn_seg < i1 + segments + 1 + 1; ++nrn_seg) {
			S->NODE_A[nrn_seg] = -1.e2 * S->NODE_RINV[nrn_seg] / S->NODE_AREA[nrn_seg - 1];
		}
		// now the effect of parent on node equation
		// todo sec->pnode needs +1 index
		for (nrn_seg = i1 + 1; nrn_seg < i1 + segments + 1 + 1; ++nrn_seg) {
			S->NODE_B[nrn_seg] = -1.e2 * S->NODE_RINV[nrn_seg] / S->NODE_AREA[nrn_seg];
		}
	}
	// for extracellular
	ext_con_coef(S, P, N);

	/**
	 * note: from LHS, this functions just recalc each time the constant NODED (!)
	 * void nrn_lhs(NrnThread *_nt)
	 * NODE_D[nrn, nd] updating is located at nrn_rhs, because _g is not the global variable
	 */
	// nt->cj = 2/dt if (secondorder) else 1/dt
	// note, the first is CAP
	// function nrn_cap_jacob(_nt, _nt->tml->ml);
	double cj = 1.0 / dt;
	double cfac = 0.001 * cj;
	for (int nrn = 0; nrn < N->size; ++nrn) {
		if (P->models[nrn] == GENERATOR)
			continue;
		i1 = P->nrn_start_seg[nrn];
		i3 = P->nrn_start_seg[nrn + 1];
		segments = (i3 - i1 - 2);
		for (nrn_seg = i1 + 1; nrn_seg < i1 + segments + 1; ++nrn_seg) {  // added + 1 for nodelist
			S->const_NODE_D[nrn_seg] += cfac * P->Cm[nrn];
		}
		// updating NODED
		for (nrn_seg = i1 + 1; nrn_seg < i3; ++nrn_seg) {
			S->const_NODE_D[nrn_seg] -= S->NODE_B[nrn_seg];
			S->const_NODE_D[nrn_seg - 1] -= S->NODE_A[nrn_seg];
		}
	}
	// extra
	// _a_matelm += NODE_A[nrn, nd]
	// _b_matelm += NODE_B[nrn, nd]
}

__global__
void initialization_kernel(States* S, Parameters* P, Neurons* N, double v_init) {
	/**
	 *
	 */
	if (blockIdx.x * blockDim.x + threadIdx.x == 0) {
		int i1, i3;
		printf("GPU: initialization_kernel\n");
		//
		connection_coef(S, P, N);
		// for different models -- different init function
		for (int nrn = 0; nrn < N->size; ++nrn) {
			// do not init neuron state for generator
			if (P->models[nrn] == GENERATOR)
				continue;
			i1 = P->nrn_start_seg[nrn];
			i3 = P->nrn_start_seg[nrn + 1];
			// for each segment init the neuron model
			for (int nrn_seg = i1; nrn_seg < i3; ++nrn_seg) {
				S->Vm[nrn_seg] = v_init;
				if (P->models[nrn] == INTER || P->models[nrn] == AFFERENTS) {
					nrn_inter_initial(S, P, N, nrn_seg, v_init);
				} else if (P->models[nrn] == MOTO) {
					nrn_moto_initial(S, P, N, nrn_seg, v_init);
				} else if (P->models[nrn] == MUSCLE) {
					nrn_muslce_initial(S, P, N, nrn_seg, v_init);
				} else {

				}
			}
			// init RHS/LHS
			setup_tree_matrix(S, P, N, nrn, i1, i3);
			// init tau synapses
			syn_initial(S, P, N, nrn);
		}
	}
}

__global__
void neuron_kernel(States *S, Parameters *P, Neurons *N, Generators *G, int t) {
	/**
	 *
	 */
	int i1, i3;
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	for (int nrn = tid; nrn < N->size; nrn += blockDim.x * gridDim.x) {
		// reset the spike state
		N->has_spike[nrn] = false;
		//
		if (P->models[nrn] != GENERATOR) {
			// calc the borders of the neuron by theirs segments
			i1 = P->nrn_start_seg[nrn];
			i3 = P->nrn_start_seg[nrn + 1];
			// re-calc currents and states based on synaptic activity
			setup_tree_matrix(S, P, N, nrn, i1, i3);
			// solve equations
			nrn_solve(S, P, N, nrn, i1, i3);
			// change voltage of the neurons based on solved equations
			update(S, P, N, nrn, i1, i3);
			// recalc conductance, update channels and deliver network events
			nrn_fixed_step_lastpart(S, P, N, nrn, i1, i3);
		}
	}
	// update generators
	if (tid == 0) {
		for (int generator = 0; generator < G->size; ++generator) {
			if (t == G->spike_each_step[generator] && t < G->time_end[generator]) {
				G->spike_each_step[generator] += G->freq_in_steps[generator];
				N->has_spike[G->nrn_id[generator]] = true;
			}
		}
	}
}

__global__
void synapse_kernel(Neurons *N, Synapses* synapses) {
	/**
	 * void deliver_net_events(NrnThread* nt)
	 */
	int pre_nrn, post_id;
	double weight;
	for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < synapses->size; index += blockDim.x * gridDim.x) {
		pre_nrn = synapses->syn_pre_nrn[index];
		// synapse update
		if (synapses->syn_delay_timer[index] > 0) {
			synapses->syn_delay_timer[index]--;
		// if timer is over -> synapse change the conductance of the post neuron
		} else if (synapses->syn_delay_timer[index] == 0) {
			post_id = synapses->syn_post_nrn[index];
			weight = synapses->syn_weight[index];
			if (weight >= 0) {
				atomicAdd(&N->g_exc[post_id], weight);
			} else {
				atomicAdd(&N->g_inh_A[post_id], -weight * N->factor[post_id]);
				atomicAdd(&N->g_inh_B[post_id], -weight * N->factor[post_id]);
			}
			synapses->syn_delay_timer[index] = -1;
		// if pre nrn has spike and synapse is ready to send siagnal
		} else if (N->has_spike[pre_nrn] && synapses->syn_delay_timer[index] == -1) {
			synapses->syn_delay_timer[index] = synapses->syn_delay[index];
		}
	}
}

void conn_generator(Group &generator, Group &post_neurons, double delay, double weight, int indegree=50) {
	/**
	 *
	 */
	uniform_int_distribution<int> nsyn_distr(indegree, indegree + 5);
	normal_distribution<double> delay_distr(delay, delay / 5);
	normal_distribution<double> weight_distr(weight, weight / 6);

	int nsyn = nsyn_distr(rand_gen);
	//
	for (int post = post_neurons.id_start; post <= post_neurons.id_end; ++post) {
		for (int i = 0; i < nsyn; ++i) {
			vector_syn_pre_nrn.push_back(generator.id_start);
			vector_syn_post_nrn.push_back(post);
			vector_syn_weight.push_back(weight_distr(rand_gen));
			vector_syn_delay.push_back(ms_to_step(delay_distr(rand_gen)));
			vector_syn_delay_timer.push_back(-1);
		}
	}
	printf("Connect generator %s [%d] to %s [%d] (1:%d). Synapses %d, D=%.1f, W=%.2f\n",
	       generator.group_name.c_str(), generator.group_size,
	       post_neurons.group_name.c_str(), post_neurons.group_size,
	       post_neurons.group_size, generator.group_size * post_neurons.group_size, delay, weight);
}

void connect_fixed_indegree(Group &pre_neurons, Group &post_neurons, double delay, double weight, int indegree=50) {
	/**
	 *
	 */
	if (vector_models[post_neurons.id_start] == INTER) {
		printf("POST INTER ");
		weight /= 10;
	}

	uniform_int_distribution<int> nsyn_distr(indegree - 15, indegree);
	uniform_int_distribution<int> pre_nrns_ids(pre_neurons.id_start, pre_neurons.id_end);
	normal_distribution<double> delay_distr(delay, delay / 5);
	normal_distribution<double> weight_distr(weight, weight / 6);
	auto nsyn = nsyn_distr(rand_gen);
	//
	for (int post = post_neurons.id_start; post <= post_neurons.id_end; ++post) {
		for (int i = 0; i < nsyn; ++i) {
			vector_syn_pre_nrn.push_back(pre_nrns_ids(rand_gen));
			vector_syn_post_nrn.push_back(post);
			vector_syn_weight.push_back(weight_distr(rand_gen));
			vector_syn_delay.push_back(ms_to_step(delay_distr(rand_gen)));
			vector_syn_delay_timer.push_back(-1);
		}
	}
	printf("Connect indegree %s [%d] to %s [%d] (%d:1). Synapses %d, D=%.1f, W=%.6f\n",
	       pre_neurons.group_name.c_str(), pre_neurons.group_size,
	       post_neurons.group_name.c_str(), post_neurons.group_size,
	       indegree, post_neurons.group_size * indegree, delay, weight);
}

void connectinsidenucleus(Group &nucleus) {
	connect_fixed_indegree(nucleus, nucleus, 0.5, 0.25);
}

void file_writing(int test_index, GroupMetadata &metadata, const string &folder) {
	/**
	 *
	 */
	ofstream file;
	string file_name = "/dat/" + to_string(test_index) + "_" + metadata.group.group_name + ".dat";

	file.open(folder + file_name);
	// save voltage
	for (unsigned int sim_iter = 0; sim_iter < SIM_TIME_IN_STEPS; sim_iter++)
		file << metadata.voltage_array[sim_iter] << " ";
	file << endl;

	// save g_exc
	for (unsigned int sim_iter = 0; sim_iter < SIM_TIME_IN_STEPS; sim_iter++)
		file << metadata.g_exc[sim_iter] << " ";
	file << endl;

	// save g_inh
	for (unsigned int sim_iter = 0; sim_iter < SIM_TIME_IN_STEPS; sim_iter++)
		file << metadata.g_inh[sim_iter] << " ";
	file << endl;

	// save spikes
	for (double const &value: metadata.spike_vector) {
		file << value << " ";
	}
	file.close();

	cout << "Saved to: " << folder + file_name << endl;
}

void save(vector<Group> groups) {
	for (Group &group : groups) {
		GroupMetadata new_meta(group, SIM_TIME_IN_STEPS);
		saving_groups.emplace_back(new_meta);
	}
}

void copy_data_to(GroupMetadata& metadata,
                  const double* Vm,
                  const double* g_exc,
                  const double* g_inh_A,
                  const double* g_inh_B,
                  const bool* has_spike,
                  const unsigned int sim_iter) {
	double nrn_mean_volt = 0;
	double nrn_mean_g_exc = 0;
	double nrn_mean_g_inh = 0;

	int center;
	for (unsigned int nrn = metadata.group.id_start; nrn <= metadata.group.id_end; ++nrn) {
		center = vector_nrn_start_seg[nrn] + ((vector_models[nrn] == MUSCLE)? 2 : 1);
		nrn_mean_volt += Vm[center];
		nrn_mean_g_exc += g_exc[nrn];
		nrn_mean_g_inh += (g_inh_B[nrn] - g_inh_A[nrn]);
		if (has_spike[nrn]) {
			metadata.spike_vector.push_back(step_to_ms(sim_iter));
		}
	}
	metadata.voltage_array[sim_iter] = nrn_mean_volt / metadata.group.group_size;
	metadata.g_exc[sim_iter] = nrn_mean_g_exc / metadata.group.group_size;
	metadata.g_inh[sim_iter] = nrn_mean_g_inh / metadata.group.group_size;
}


void save_result(int test_index) {
	string current_path = getcwd(nullptr, 0);

	printf("[Test #%d] Save results to: %s \n", test_index, current_path.c_str());

	for (GroupMetadata &metadata : saving_groups) {
		file_writing(test_index, metadata, current_path);
	}
}

template<typename type>
type* arr_segs() {
	// important: nrns_and_segs initialized at network building
	return new type[nrns_and_segs]();
}

void createmotif(Group OM0, Group OM1, Group OM2, Group OM3) {
	/**
	 * Connects motif module
	 * see https://github.com/research-team/memristive-spinal-cord/blob/master/doc/diagram/cpg_generator_FE_paper.png
	 */
	connect_fixed_indegree(OM0, OM1, 3, 2.85);
	connect_fixed_indegree(OM1, OM2, 3, 2.85);
	connect_fixed_indegree(OM2, OM1, 3, 1.95);
	connect_fixed_indegree(OM2, OM3, 3, 0.0005);
	connect_fixed_indegree(OM1, OM3, 3, 0.00005);
	connect_fixed_indegree(OM3, OM2, 3, -4.5);
	connect_fixed_indegree(OM3, OM1, 3, -4.5);
}

void init_network() {
	/**
	 * todo
	 */
	string name;
	vector<Group> CV, CV_1, L0, L1, L2E, L2F, L3, IP_E, IP_F, gen_C, C_0, V0v;
	// generators
	auto ees = form_group("EES", 1, GENERATOR);
	for(int layer = 0; layer < layers + 1; ++layer) {
		name = to_string(layer + 1);
		gen_C.push_back(form_group("C" + name, 1, GENERATOR));
	}
	for(int step = 0; step < step_number; ++step) {
		name = to_string(step);
		C_0.push_back(form_group("C_0_step_" + name, 1, GENERATOR));
		V0v.push_back(form_group("V0v_step_" + name, 1, GENERATOR));
	}
	//
	auto OM1_0E = form_group("OM1_0E");
	auto OM1_0F = form_group("OM1_0F");
	// OM groups by layer
	for(int layer = 0; layer < layers; ++layer) {
		name = to_string(layer + 1);
		L0.push_back(form_group("OM" + name + "_0"));
		L1.push_back(form_group("OM" + name + "_1"));
		L2E.push_back(form_group("OM" + name + "_2E"));
		L2F.push_back(form_group("OM" + name + "_2F"));
		L3.push_back(form_group("OM" + name + "_3"));
	}
	//
	for(int layer = 0; layer < layers + 1; ++layer) {
		name = to_string(layer + 1);
		CV.push_back(form_group("CV" + name, 50, AFFERENTS));        // E-шки
		CV_1.push_back(form_group("CV_1_" + name, 50, AFFERENTS));   // true CV
		// interneuronal pool
		IP_E.push_back(form_group("IP_E_" + name));
		IP_F.push_back(form_group("IP_F_" + name));
	}
	// afferents
	auto Ia_aff_E = form_group("Ia_aff_E", 120, AFFERENTS);
	auto Ia_aff_F = form_group("Ia_aff_F", 120, AFFERENTS);
	// motoneurons
	auto mns_E = form_group("mns_E", 210, MOTO);
	auto mns_F = form_group("mns_F", 180, MOTO);
	// muscle fibers
	auto muscle_E = form_group("muscle_E", 20, MUSCLE, 3); // 150 * 210
	auto muscle_F = form_group("muscle_F", 20, MUSCLE, 3); // 100 * 180
	// reflex arc E
	auto Ia_E = form_group("Ia_E", neurons_in_ip);
	auto iIP_E = form_group("iIP_E", neurons_in_ip);
	auto R_E = form_group("R_E");
	// reflex arc F
	auto Ia_F = form_group("Ia_F", neurons_in_ip);
	auto iIP_F = form_group("iIP_F", neurons_in_ip);
	auto R_F = form_group("R_F");

	// note: must be at the end of a group forming
	vector_nrn_start_seg.push_back(nrns_and_segs);

	// create generators
	add_generator(ees, 0, sim_time, ees_fr);
	for (int layer = 0; layer < layers + 1; ++layer) {
		for (int step_index = 0; step_index < step_number; ++step_index) {
			normal_distribution<double> freq_distr(cv_fr, cv_fr / 10);
			double start = 25 + skin_time * layer + step_index * (skin_time * (layers + 1) + flexor_dur);
			double end = start + skin_time;
			add_generator(gen_C[layer], start, end, freq_distr(rand_gen));
		}
	}
	//
	for (int step_index = 0; step_index < step_number; ++step_index) {
		// freq = 200 (interval = 5ms), count = 125 / interval. Duration = count * interval = 125
		double start = 25 + skin_time * 6 + step_index * (skin_time * 6 + flexor_dur);
		double end = start + 125;
		add_generator(C_0[step_index], start, end, cv_fr);
		// V0v
		start = 40 + skin_time * 6 + step_index * (skin_time * 6 + flexor_dur);
		end = start + 75;
		add_generator(V0v[step_index], start, end, cv_fr);
	}
	// extensor
	createmotif(OM1_0E, L1[0], L2E[0], L3[0]);
	for(int layer = 1; layer < layers; ++layer)
		createmotif(L0[layer], L1[layer], L2E[layer], L3[layer]);
	// extra flexor connections
	createmotif(OM1_0F, L1[0], L2E[0], L3[0]);
	for(int layer = 1; layer < layers; ++layer)
		createmotif(L0[layer], L1[layer], L2F[layer], L3[layer]);

	for(int layer = 1; layer < layers; ++layer)
		connect_fixed_indegree(L2F[layer - 1], L2F[layer], 2, 1.5);
	//
	connect_fixed_indegree(CV[0], OM1_0F, 3, 0.0005);
	for(int step = 0; step < step_number; ++step) {
		connect_fixed_indegree(V0v[step], OM1_0F, 3, 2.75);
	}
	// between delays via excitatory pools
	// extensor
	for(int layer = 1; layer < layers; ++layer)
		connect_fixed_indegree(CV[layer - 1], CV[layer], 3, 2);
	// connect E (from EES)
	connect_fixed_indegree(CV[0], OM1_0E, 2, 0.00027); // 0.00047
	for(int layer = 1; layer < layers; ++layer)
		connect_fixed_indegree(CV[layer], L0[layer], 2, 0.00028); // 0.00048

	// CV inhibitory projections (via 3rd core)
	for (int layer = 0; layer < layers - 1; ++layer) {
		if (layer >= 3) {
			for (int i = layer + 3; i < layers + 1; ++i) {
				printf("C index %d, OM%d_3 (layer > 3)\n", i, layer);
				connect_fixed_indegree(gen_C[i], L3[layer], 1, 1.95);
			}
		} else {
			for (int i = layer + 2; i < layers + 1; ++i) {
				printf("C index %d, OM%d_3 (else)\n", i, layer);
				connect_fixed_indegree(gen_C[i], L3[layer], 1, 1.95);
			}
		}
	}

	conn_generator(ees, Ia_aff_E, 1, 1.5);
	conn_generator(ees, Ia_aff_F, 1, 1.5);
	conn_generator(ees, CV[0], 2, 1.5);
	///conn_generator(Iagener_E, Ia_aff_E, 1, 0.0001, 5);
	///conn_generator(Iagener_F, Ia_aff_F, 1, 0.0001, 5);

	connect_fixed_indegree(Ia_aff_E, mns_E, 1.5, 1.55);
	connect_fixed_indegree(Ia_aff_F, mns_F, 1.5, 1.5);

	connect_fixed_indegree(mns_E, muscle_E, 2, 15.5, 45);
	connect_fixed_indegree(mns_F, muscle_F, 2, 15.5, 45);
	// IP
	for (int layer = 0; layer < layers; ++layer) {
		// Extensor
		connectinsidenucleus(IP_F[layer]);
		connectinsidenucleus(L2E[layer]);
		connectinsidenucleus(L2F[layer]);
		connect_fixed_indegree(L2E[layer], IP_E[layer], 3, 2.85);
		connect_fixed_indegree(IP_E[layer], mns_E, 3, 2.85);
		if (layer > 3)
			connect_fixed_indegree(IP_E[layer], Ia_aff_E, 1, -layer * 0.0002);
		else
			connect_fixed_indegree(IP_E[layer], Ia_aff_E, 1, -0.0001);
		// Flexor
		connect_fixed_indegree(L2F[layer], IP_F[layer], 3, 3.5);
		connect_fixed_indegree(IP_F[layer], mns_F, 2, 3.5);
		connect_fixed_indegree(IP_F[layer], Ia_aff_F, 1, -0.85);
	}
	// skin inputs
	for (int layer = 0; layer < layers + 1; ++layer)
		connect_fixed_indegree(gen_C[layer], CV_1[layer], 2, 0.15 * k * skin_time);

	// C
	// C1
	connect_fixed_indegree(CV_1[0], OM1_0E, 2, 0.00075 * k * skin_time);
	connect_fixed_indegree(CV_1[0], L0[1], 3, 0.00001 * k * skin_time);
	connect_fixed_indegree(CV_1[0], L0[2], 3, 0.00001 * k * skin_time);
    // C2
	connect_fixed_indegree(CV_1[1], OM1_0E, 2, 0.0005 * k * skin_time);
	connect_fixed_indegree(CV_1[1], L0[1], 3, 0.00045 * k * skin_time);
	connect_fixed_indegree(CV_1[1], L0[2], 3, 0.00025 * k * skin_time);
	connect_fixed_indegree(CV_1[1], L0[3], 3, 0.00005 * k * skin_time);
    // C3
	connect_fixed_indegree(CV_1[2], L0[1], 2, 0.0004 * k * skin_time);
	connect_fixed_indegree(CV_1[2], L0[2], 3, 0.00035 * k * skin_time);
	connect_fixed_indegree(CV_1[2], L0[3], 3, 0.0002 * k * skin_time);
	connect_fixed_indegree(CV_1[2], L0[4], 3, 0.0001 * k * skin_time);
    // C4
	connect_fixed_indegree(CV_1[3], L0[2], 3, 0.00035 * k * skin_time);
	connect_fixed_indegree(CV_1[3], L0[3], 3, 0.00035 * k * skin_time);
	connect_fixed_indegree(CV_1[4], L0[2], 3, 0.00035 * k * skin_time);
	connect_fixed_indegree(CV_1[4], L0[3], 3, 0.00035 * k * skin_time);
	connect_fixed_indegree(CV_1[3], L0[4], 3, 0.0001 * k * skin_time);
	connect_fixed_indegree(CV_1[4], L0[4], 3, 0.0001 * k * skin_time);
	// C5
	connect_fixed_indegree(CV_1[5], L0[4], 3, 0.00025 * k * skin_time);
	connect_fixed_indegree(CV_1[5], L0[3], 3, 0.0001 * k * skin_time);
	// C=1 Extensor
	for (int layer = 0; layer < layers; ++layer)
		connect_fixed_indegree(IP_E[layer], iIP_E, 1, 0.001);
	//
	for (int layer = 0; layer < layers + 1; ++layer) {
		connect_fixed_indegree(CV_1[layer], iIP_E, 1, 1.8);
		connect_fixed_indegree(gen_C[layer], iIP_E, 1, 1.8);
	}
	connect_fixed_indegree(iIP_E, OM1_0F, 1, -1.9);

	for (int layer = 0; layer < layers; ++layer) {
		connect_fixed_indegree(iIP_E, L2F[layer], 2, -1.8);
		connect_fixed_indegree(iIP_F, L2E[layer], 2, -0.5);
	}
	//
	connect_fixed_indegree(iIP_E, Ia_aff_F, 1, -1.2);
	connect_fixed_indegree(iIP_E, mns_F, 1, -0.8);
	for (int layer = 0; layer < layers; ++layer) {
		connect_fixed_indegree(iIP_E, IP_F[layer], 1, -0.5);
		connect_fixed_indegree(IP_F[layer], iIP_F, 1, 0.0001);
		connect_fixed_indegree(iIP_F, IP_E[layer], 1, -0.8);
	}
	// C=0 Flexor
	connect_fixed_indegree(iIP_F, iIP_E, 1, -0.5);
	connect_fixed_indegree(iIP_F, Ia_aff_E, 1, -0.5);
	connect_fixed_indegree(iIP_F, mns_E, 1, -0.4);
	for(int step = 0; step < step_number; ++step) {
		connect_fixed_indegree(C_0[step], iIP_F, 1, 0.8);
	}
	// reflex arc
	connect_fixed_indegree(iIP_E, Ia_E, 1, 0.001);
	connect_fixed_indegree(Ia_aff_E, Ia_E, 1, 0.008);
	connect_fixed_indegree(mns_E, R_E, 1, 0.00015);
	connect_fixed_indegree(Ia_E, mns_F, 1, -0.08);
	connect_fixed_indegree(R_E, mns_E, 1, -0.00015);
	connect_fixed_indegree(R_E, Ia_E, 1, -0.001);
	//
	connect_fixed_indegree(iIP_F, Ia_F, 1, 0.001);
	connect_fixed_indegree(Ia_aff_F, Ia_F, 1, 0.008);
	connect_fixed_indegree(mns_F, R_F, 1, 0.00015);
	connect_fixed_indegree(Ia_F, mns_E, 1, -0.08);
	connect_fixed_indegree(R_F, mns_F, 1, -0.00015);
	connect_fixed_indegree(R_F, Ia_F, 1, -0.001);
	//
	connect_fixed_indegree(R_E, R_F, 1, -0.04);
	connect_fixed_indegree(R_F, R_E, 1, -0.04);
	connect_fixed_indegree(Ia_E, Ia_F, 1, -0.08);
	connect_fixed_indegree(Ia_F, Ia_E, 1, -0.08);
	connect_fixed_indegree(iIP_E, iIP_F, 1, -0.04);
	connect_fixed_indegree(iIP_F, iIP_E, 1, -0.04);
	//
//	vector<Group> groups = {L0[0], L1[0], L3[0], Ia_aff_E, gen_C[0], ees, CV[0], OM1_0E};
//	save(groups);
	save(all_groups);
}

void simulate(int test_index) {
	/**
	 *
	 */
	// init structs (CPU)
	States *S = (States *)malloc(sizeof(States));
	Parameters *P = (Parameters *)malloc(sizeof(Parameters));
	Neurons *N = (Neurons *)malloc(sizeof(Neurons));
	Synapses *synapses = (Synapses *)malloc(sizeof(Synapses));
	Generators *G = (Generators *)malloc(sizeof(Generators));

	// create neurons and their connectomes
	init_network();

	// allocate generators into the GPU
	unsigned int gens_number = vec_spike_each_step.size();
	G->nrn_id = init_gpu_arr(vec_nrn_id);
	G->time_end = init_gpu_arr(vec_time_end);
	G->freq_in_steps = init_gpu_arr(vec_freq_in_steps);
	G->spike_each_step = init_gpu_arr(vec_spike_each_step);
	G->size = gens_number;

	// allocate static parameters into the GPU
	P->nrn_start_seg = init_gpu_arr(vector_nrn_start_seg);
	P->models = init_gpu_arr(vector_models);
	P->Cm = init_gpu_arr(vector_Cm);
	P->gnabar = init_gpu_arr(vector_gnabar);
	P->gkbar = init_gpu_arr(vector_gkbar);
	P->gl = init_gpu_arr(vector_gl);
	P->Ra = init_gpu_arr(vector_Ra);
	P->diam = init_gpu_arr(vector_diam);
	P->length = init_gpu_arr(vector_length);
	P->ena = init_gpu_arr(vector_ena);
	P->ek = init_gpu_arr(vector_ek);
	P->el = init_gpu_arr(vector_el);
	P->gkrect = init_gpu_arr(vector_gkrect);
	P->gcaN = init_gpu_arr(vector_gcaN);
	P->gcaL = init_gpu_arr(vector_gcaL);
	P->gcak = init_gpu_arr(vector_gcak);
	P->E_ex = init_gpu_arr(vector_E_ex);
	P->E_inh = init_gpu_arr(vector_E_inh);
	P->tau_exc = init_gpu_arr(vector_tau_exc);
	P->tau_inh1 = init_gpu_arr(vector_tau_inh1);
	P->tau_inh2 = init_gpu_arr(vector_tau_inh2);
	P->size = nrns_number;

	// dynamic states of neuron (CPU arrays) and allocate them into the GPU
	auto *Vm = arr_segs<double>(); S->Vm = init_gpu_arr(Vm);
	auto *n = arr_segs<double>(); S->n = init_gpu_arr(n);
	auto *m = arr_segs<double>(); S->m = init_gpu_arr(m);
	auto *h = arr_segs<double>(); S->h = init_gpu_arr(h);
	auto *l = arr_segs<double>(); S->l = init_gpu_arr(l);
	auto *s = arr_segs<double>(); S->s = init_gpu_arr(s);
	auto *p = arr_segs<double>(); S->p = init_gpu_arr(p);
	auto *hc = arr_segs<double>(); S->hc = init_gpu_arr(hc);
	auto *mc = arr_segs<double>(); S->mc = init_gpu_arr(mc);
	auto *cai = arr_segs<double>(); S->cai = init_gpu_arr(cai);
	auto *I_Ca = arr_segs<double>(); S->I_Ca = init_gpu_arr(I_Ca);
	auto *NODE_A = arr_segs<double>(); S->NODE_A = init_gpu_arr(NODE_A);
	auto *NODE_B = arr_segs<double>(); S->NODE_B = init_gpu_arr(NODE_B);
	auto *NODE_D = arr_segs<double>(); S->NODE_D = init_gpu_arr(NODE_D);
	auto *const_NODE_D = arr_segs<double>(); S->const_NODE_D = init_gpu_arr(const_NODE_D);
	auto *NODE_RHS = arr_segs<double>(); S->NODE_RHS = init_gpu_arr(NODE_RHS);
	auto *NODE_RINV = arr_segs<double>(); S->NODE_RINV = init_gpu_arr(NODE_RINV);
	auto *NODE_AREA = arr_segs<double>(); S->NODE_AREA = init_gpu_arr(NODE_AREA);
	S->size = nrns_and_segs;

	// special neuron's state (CPU) and allocate them into the GPU
	auto *has_spike = arr_segs<bool>(); N->has_spike = init_gpu_arr(has_spike);
	auto *spike_on = arr_segs<bool>(); N->spike_on = init_gpu_arr(spike_on);
	auto *g_exc = arr_segs<double>(); N->g_exc = init_gpu_arr(g_exc);
	auto *g_inh_A = arr_segs<double>(); N->g_inh_A = init_gpu_arr(g_inh_A);
	auto *g_inh_B = arr_segs<double>(); N->g_inh_B = init_gpu_arr(g_inh_B);
	auto *factor = arr_segs<double>(); N->factor = init_gpu_arr(factor);
	N->size = nrns_number;

	// synaptic parameters
	unsigned int synapses_number = vector_syn_delay.size();
	synapses->syn_pre_nrn = init_gpu_arr(vector_syn_pre_nrn);
	synapses->syn_post_nrn = init_gpu_arr(vector_syn_post_nrn);
	synapses->syn_weight = init_gpu_arr(vector_syn_weight);
	synapses->syn_delay = init_gpu_arr(vector_syn_delay);
	synapses->syn_delay_timer = init_gpu_arr(vector_syn_delay_timer);
	synapses->size = synapses_number;

	// allocate structs to the device
	auto *dev_S = init_gpu_arr(S, 1);
	auto *dev_P = init_gpu_arr(P, 1);
	auto *dev_N = init_gpu_arr(N, 1);
	auto *dev_G = init_gpu_arr(G, 1);
	auto *dev_synapses = init_gpu_arr(synapses, 1);

	printf("Network: %d neurons (with segs: %d), %d synapses, %d generators\n",
	       nrns_number, nrns_and_segs, synapses_number, gens_number);

	float time;
	hipEvent_t start, stop;
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventRecord(start, 0));

	// call initialisation kernel
	initialization_kernel<<<1, 1>>>(dev_S, dev_P, dev_N, -70.0);

	// the main simulation loop
	for (unsigned int sim_iter = 0; sim_iter < SIM_TIME_IN_STEPS; ++sim_iter) {
		/// KERNEL ZONE
		// deliver_net_events, synapse updating and neuron conductance changing kernel
		synapse_kernel<<<5, 256>>>(dev_N, dev_synapses);
		// updating neurons kernel
		neuron_kernel<<<10, 32>>>(dev_S, dev_P, dev_N, dev_G, sim_iter);
		/// SAVE DATA ZONE
		memcpyDtH(S->Vm, Vm, nrns_and_segs);
		memcpyDtH(N->g_exc, g_exc, nrns_number);
		memcpyDtH(N->g_inh_A, g_inh_A, nrns_number);
		memcpyDtH(N->g_inh_B, g_inh_B, nrns_number);
		memcpyDtH(N->has_spike, has_spike, nrns_number);
		// fill records arrays
		for (GroupMetadata& metadata : saving_groups) {
			copy_data_to(metadata, Vm, g_exc, g_inh_A, g_inh_B, has_spike, sim_iter);
		}
	}
	// properly ending work with GPU
	HANDLE_ERROR(hipDeviceSynchronize());
	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	HANDLE_ERROR(hipEventElapsedTime(&time, start, stop));
	// todo optimize the code to free all GPU variables
	HANDLE_ERROR(hipFree(S->Vm));

	// stuff info
	printf("Elapsed GPU time: %d ms\n", (int) time);
	double Tbw = 12000 * pow(10, 6) * (128 / 8) * 2 / pow(10, 9);
	printf("Theoretical Bandwidth GPU (2 Ghz, 128 bit): %.2f GB/s\n", Tbw);

	// save the data into the current folder
	save_result(test_index);
}

int main(int argc, char **argv) {
	// init the device
	int dev = 0;
	hipDeviceProp_t deviceProp;
	HANDLE_ERROR(hipGetDeviceProperties(&deviceProp, dev));
	printf("%s struct of array at ", argv[0]);
	printf("device %d: %s \n", dev, deviceProp.name);
	HANDLE_ERROR(hipSetDevice(dev));
	// the main body of simulation
	simulate(0);
	// reset device
	HANDLE_ERROR(hipDeviceReset());
}
