#include "hip/hip_runtime.h"
/**
Formulas and value units were taken from:

Sterratt, D., Graham, B., Gillies, A., & Willshaw, D. (2011).
Principles of Computational Modelling in Neuroscience. Cambridge: Cambridge University Press.
DOI:10.1017/CBO9780511975899

Based on the NEURON repository
*/
#include <algorithm>
#include <utility>
#include <ctime>
#include <random>
#include <map>
#include <vector>
#include <string>
#include <iostream>
#include "test.h"
#include <stdexcept>
// for file writing
#include <cstdlib>
#include <iostream>
#include <fstream>
#include <unistd.h>
#define HANDLE_ERROR( err ) ( HandleError( err, __FILE__, __LINE__ ) )
#define PI 3.141592654f

static void HandleError(hipError_t err, const char *file, int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit(EXIT_FAILURE);
	}
}

using namespace std;

const float dt = 0.025;      // [ms] simulation step
const int sim_time = 50;    // [ms] simulation time
const auto SIM_TIME_IN_STEPS = (unsigned int)(sim_time / dt);  // [steps] converted time into steps

const bool DEBUG = false;
const bool EXTRACELLULAR = false;
const char GENERATOR = 'g';
const char INTER = 'i';
const char MOTO = 'm';
const char MUSCLE = 'u';

const int skin_time = 25;  // duration of layer 25 = 21 cm/s; 50 = 15 cm/s; 125 = 6 cm/s
int cv_fr = 200;     // frequency of CV
int ees_fr = 100;     // frequency of EES

float cv_int = 1000 / cv_fr;
float ees_int = 1000 / ees_fr;

random_device r;
default_random_engine generator(r());

unsigned int nrns_number = 0;        // [id] global neuron id = number of neurons
unsigned int nrns_and_segs = 0;      // [id] global neuron+segs id = number of neurons with segments
unsigned int generators_id_end = 0;  // [id] id of the last generator (to avoid them for updating)
const int LEG_STEPS = 1;             // [step] number of full cycle steps

const int neurons_in_group = 50;     // number of neurons in a group
const int neurons_in_ip = 196;       // number of neurons in a group


/*
EES_stimulus = (np.arange(0, sim_time, ees_int) / dt).astype(int)
CV1_stimulus = (np.arange(skin_time * 0, skin_time * 1, random.gauss(cv_int, cv_int / 10)) / dt).astype(int)
CV2_stimulus = (np.arange(skin_time * 1, skin_time * 2, random.gauss(cv_int, cv_int / 10)) / dt).astype(int)
CV3_stimulus = (np.arange(skin_time * 2, skin_time * 3, random.gauss(cv_int, cv_int / 10)) / dt).astype(int)
CV4_stimulus = (np.arange(skin_time * 3, skin_time * 5, random.gauss(cv_int, cv_int / 10)) / dt).astype(int)
CV5_stimulus = (np.arange(skin_time * 5, skin_time * 6, random.gauss(cv_int, cv_int / 10)) / dt).astype(int)
*/

/*
# arrays for saving data
spikes = []             # saved spikes
GRAS_data = []          # saved gras data (DEBUGGING)
save_groups = []        # neurons groups that need to save
saved_voltage = []      # saved voltage
save_neuron_ids = []    # neurons id that need to save
 */

// common neuron constants
const float k = 0.017;           // synaptic coef
const float V_th = -40;          // [mV] voltage threshold
const float V_adj = -63;         // [mV] adjust voltage for -55 threshold
// moto neuron constants
const float ca0 = 2;             // initial calcium concentration
const float amA = 0.4;           // const ??? todo
const float amB = 66;            // const ??? todo
const float amC = 5;             // const ??? todo
const float bmA = 0.4;           // const ??? todo
const float bmB = 32;            // const ??? todo
const float bmC = 5;             // const ??? todo
const float R_const = 8.314472;  // [k-mole] or [joule/degC] const
const float F_const = 96485.34;  // [faraday] or [kilocoulombs] const
// muscle fiber constants
const float g_kno = 0.01;        // [S/cm2] conductance of the todo
const float g_kir = 0.03;        // [S/cm2] conductance of the Inwardly Rectifying Potassium K+ (Kir) channel
// Boltzman steady state curve
const float vhalfl = -98.92;     // [mV] inactivation half-potential
const float kl = 10.89;          // [mV] Stegen et al. 2012
// tau_infty
const float vhalft = 67.0828;    // [mV] fitted //100 uM sens curr 350a, Stegen et al. 2012
const float at = 0.00610779;     // [/ ms] Stegen et al. 2012
const float bt = 0.0817741;      // [/ ms] Note: typo in Stegen et al. 2012
// temperature dependence
const float q10 = 1;             // temperature scaling (sensitivity)
const float celsius = 36;        // [degC] temperature of the cell
// i_membrane [mA/cm2]
const float e_extracellular = 0; // [mV]
const float xraxial = 1e9;       // [MOhm/cm]

// Allocate and fill host data
vector<short> vector_nrn_start_seg;
vector<char> vector_models;
vector<float> vector_Cm;
vector<float> vector_gnabar;
vector<float> vector_gkbar;
vector<float> vector_gl;
vector<float> vector_Ra;
vector<float> vector_diam;
vector<float> vector_length;
vector<float> vector_ena;
vector<float> vector_ek;
vector<float> vector_el;
vector<float> vector_gkrect;
vector<float> vector_gcaN;
vector<float> vector_gcaL;
vector<float> vector_gcak;
vector<float> vector_E_ex;
vector<float> vector_E_inh;
vector<float> vector_tau_exc;
vector<float> vector_tau_inh1;
vector<float> vector_tau_inh2;
// synapses
vector<int> vector_syn_pre_nrn;       // [id] list of pre neurons ids
vector<int> vector_syn_post_nrn;      // [id] list of pre neurons ids
vector<float> vector_syn_weight;      // [S] list of synaptic weights
vector<int> vector_syn_delay;         // [ms * dt] list of synaptic delays in steps
vector<int> vector_syn_delay_timer;   // [ms * dt] list of synaptic timers, shows how much left to send signal
// results vector
vector <GroupMetadata> saving_groups;    //

// form structs of neurons global ID and groups name
Group form_group(const string &group_name,
				 int nrns_in_group = neurons_in_group,
				 const char model = INTER,
				 const int segs = 1) {
	Group group = Group();
	group.group_name = group_name;     // name of a neurons group
	group.id_start = nrns_number;        // first ID in the group
	group.id_end = nrns_number + nrns_in_group - 1;  // the latest ID in the group
	group.group_size = nrns_in_group;  // size of the neurons group

	float __Cm;
	float __gnabar;
	float __gkbar;
	float __gl;
	float __Ra;
	float __ena;
	float __ek;
	float __el;
	float __diam;
	float __dx;
	float __gkrect;
	float __gcaN;
	float __gcaL;
	float __gcak;
	float __e_ex;
	float __e_inh;
	float __tau_exc;
	float __tau_inh1;
	float __tau_inh2;
	normal_distribution<float> Cm_distr(1, 0.01);
	uniform_real_distribution<float> moto_diam_distr(45, 55);

	for (int nrn = 0; nrn < nrns_in_group; nrn++) {
		if (model == INTER) {
			__Cm = Cm_distr(generator);
			__gnabar = 0.1;
			__gkbar = 0.08;
			__gl = 0.002;
			__Ra = 100.0;
			__ena = 50.0;
			__ek = -90.0;
			__el = -70.0;
			__diam = 10.0; // random.randint(5, 15);
			__dx = __diam;
			__e_ex = 50;
			__e_inh = -80;
			__tau_exc = 0.35;
			__tau_inh1 = 0.5;
			__tau_inh2 = 3.5;
		} else if (model == MOTO) {
			__Cm = 2;
			__gnabar = 0.05;
			__gl = 0.002;
			__Ra = 200.0;
			__ena = 50.0;
			__ek = -80.0;
			__el = -70.0;
			__diam = moto_diam_distr(generator); //random.randint(45, 55);
			__dx = __diam;
			__gkrect = 0.3;
			__gcaN = 0.05;
			__gcaL = 0.0001;
			__gcak = 0.3;
			__e_ex = 50.0;
			__e_inh = -80.0;
			__tau_exc = 0.3;
			__tau_inh1 = 1.0;
			__tau_inh2 = 1.5;
			if (__diam > 50) {
				__gnabar = 0.1;
				__gcaL = 0.001;
				__gl = 0.003;
				__gkrect = 0.2;
				__gcak = 0.2;
			}
		} else if (model == MUSCLE) {
			__Cm = 3.6;
			__gnabar = 0.15;
			__gkbar = 0.03;
			__gl = 0.0002;
			__Ra = 1.1;
			__ena = 55.0;
			__ek = -80.0;
			__el = -72.0;
			__diam = 40.0;
			__dx = 3000.0;
			__e_ex = 0.0;
			__e_inh = -80.0;
			__tau_exc = 0.3;
			__tau_inh1 = 1.0;
			__tau_inh2 = 1.0;
		} else if (model == GENERATOR) {

		} else {
			throw logic_error("Choose the model");
		}
		// common properties
		vector_Cm.push_back(__Cm);
		vector_gnabar.push_back(__gnabar);
		vector_gkbar.push_back(__gkbar);
		vector_gl.push_back(__gl);
		vector_el.push_back(__el);
		vector_ena.push_back(__ena);
		vector_ek.push_back(__ek);
		vector_Ra.push_back(__Ra);
		vector_diam.push_back(__diam);
		vector_length.push_back(__dx);
		vector_gkrect.push_back(__gkrect);
		vector_gcaN.push_back(__gcaN);
		vector_gcaL.push_back(__gcaL);
		vector_gcak.push_back(__gcak);
		vector_E_ex.push_back(__e_ex);
		vector_E_inh.push_back(__e_inh);
		vector_tau_exc.push_back(__tau_exc);
		vector_tau_inh1.push_back(__tau_inh1);
		vector_tau_inh2.push_back(__tau_inh2);
		//
		vector_nrn_start_seg.push_back(nrns_and_segs);
		nrns_and_segs += (segs + 2);
		vector_models.push_back(model);
	}

	nrns_number += nrns_in_group;
	printf("Formed %s IDs [%d ... %d] = %d\n",
		group_name.c_str(), nrns_number - nrns_in_group, nrns_number - 1, nrns_in_group);

	return group;
}

__host__
unsigned int ms_to_step(float ms) { return (unsigned int) (ms / dt); }

__host__
float step_to_ms(int step) { return step * dt; }

// copy data from host to device
template<typename type>
void memcpyHtD(type *gpu, type *host, unsigned int size) {
	hipMemcpy(gpu, host, sizeof(type) * size, hipMemcpyHostToDevice);
}

// copy data from device to host
template<typename type>
void memcpyDtH(type *host, type *gpu, unsigned int size) {
	hipMemcpy(host, gpu, size * sizeof(type), hipMemcpyDeviceToHost);
}

template<typename type>
type* init_gpu_arr(type *cpu_var, int size) {
	type *gpu_var;
	hipMalloc(&gpu_var, size * sizeof(type));
	memcpyHtD<type>(gpu_var, cpu_var, size);
	return gpu_var;
}

template<typename type>
type *init_gpu_arr(vector<type> &vec) {
	type *gpu_var;
	hipMalloc(&gpu_var, sizeof(type) * vec.size());
	memcpyHtD<type>(gpu_var, vec.data(), vec.size());
	return gpu_var;
}

template<typename type>
type* vec2arr(vector<type> &vec) {
	return vec.cpu_vector.data();
}
__device__
float Exp(float volt) {
	return (volt < -100)? 0 : exp(volt);
}

__device__
float alpham(float volt) {
	if (abs((volt + amB) / amC) < 1e-6)
		return amA * amC;
	return amA * (volt + amB) / (1.0 - Exp(-(volt + amB) / amC));
}

__device__
float betam(float volt) {
	if (abs((volt + bmB) / bmC) < 1e-6)
		return -bmA * bmC;
	return -bmA * (volt + bmB) / (1.0 - Exp((volt + bmB) / bmC));
}

__device__
float syn_current(Neurons* U, Parameters* P, int nrn, float voltage) {
	/**
	calculate synaptic current
	*/
	return U->g_exc[nrn] * (voltage - P->E_ex[nrn]) + (U->g_inh_B[nrn] - U->g_inh_A[nrn]) * (voltage - P->E_inh[nrn]);
}

__device__
float nrn_moto_current(States* S, Parameters* P, Neurons* U, int nrn, int nrn_seg_index, float voltage) {
	/**
	calculate channels current
	*/
	float iNa = P->gnabar[nrn] * pow(S->m[nrn_seg_index], 3) * S->h[nrn_seg_index] * (voltage - P->ena[nrn]);
	float iK = P->gkrect[nrn] * pow(S->n[nrn_seg_index], 4) * (voltage - P->ek[nrn]) +
               P->gcak[nrn] * pow(S->cai[nrn_seg_index], 2) / (pow(S->cai[nrn_seg_index], 2) + 0.014 * 0.014) * (voltage - P->ek[nrn]);
	float iL = P->gl[nrn] * (voltage - P->el[nrn]);
	float eCa = (1000 * R_const * 309.15 / (2 * F_const)) * log(ca0 / S->cai[nrn_seg_index]);
	S->I_Ca[nrn_seg_index] = P->gcaN[nrn] * pow(S->mc[nrn_seg_index], 2) * S->hc[nrn_seg_index] * (voltage - eCa) +
	                         P->gcaL[nrn] * S->p[nrn_seg_index] * (voltage - eCa);
	return iNa + iK + iL + S->I_Ca[nrn_seg_index];
}

__device__
float nrn_fastchannel_current(States* S, Parameters* P, Neurons* U, int nrn, int nrn_seg_index, float voltage) {
	/**
	calculate channels current
	*/
	float iNa = P->gnabar[nrn] * pow(S->m[nrn_seg_index], 3) * S->h[nrn_seg_index] * (voltage - P->ena[nrn]);
	float iK = P->gkbar[nrn] * pow(S->n[nrn_seg_index], 4) * (voltage - P->ek[nrn]);
	float iL = P->gl[nrn] * (voltage - P->el[nrn]);
	return iNa + iK + iL;
}

__device__
void recalc_synaptic(States* S, Parameters* P, Neurons* U, int nrn) {
	/**
	updating conductance(summed) of neurons' post-synaptic conenctions
	*/
	// exc synaptic conductance
	if (U->g_exc[nrn] != 0) {
		if (nrn == 10)
			printf("U->g_exc[nrn] = %f\n",  U->g_exc[nrn]);
		U->g_exc[nrn] -= (1.0 - exp(-dt / P->tau_exc[nrn])) * U->g_exc[nrn];
		if (U->g_exc[nrn] < 1e-5) {
			U->g_exc[nrn] = 0.0;
		}
	}
	// inh1 synaptic conductance
	if (U->g_inh_A[nrn] != 0) {
		U->g_inh_A[nrn] -= (1.0 - exp(-dt / P->tau_inh1[nrn])) * U->g_inh_A[nrn];
		if (U->g_inh_A[nrn] < 1e-5) {
			U->g_inh_A[nrn] = 0.0;
		}
	}
	// inh2 synaptic conductance
	if (U->g_inh_B[nrn] != 0) {
		U->g_inh_B[nrn] -= (1.0 - exp(-dt / P->tau_inh2[nrn])) * U->g_inh_B[nrn];
		if (U->g_inh_B[nrn] < 1e-5)
			U->g_inh_B[nrn] = 0.0;
	}
}

__device__
void syn_initial(States* S, Parameters* P, Neurons* U, int nrn) {
	/**
	initialize tau(rise / decay time, ms) and factor(const) variables
	*/
	if (P->tau_inh1[nrn] / P->tau_inh2[nrn] > 0.9999)
		P->tau_inh1[nrn] = 0.9999 * P->tau_inh2[nrn];
	if (P->tau_inh1[nrn] / P->tau_inh2[nrn] < 1e-9)
		P->tau_inh1[nrn] = P->tau_inh2[nrn] * 1e-9;
	//
	float tp = (P->tau_inh1[nrn] * P->tau_inh2[nrn]) / (P->tau_inh2[nrn] - P->tau_inh1[nrn]) *
	           log(P->tau_inh2[nrn] / P->tau_inh1[nrn]);
	U->factor[nrn] = -exp(-tp / P->tau_inh1[nrn]) + exp(-tp / P->tau_inh2[nrn]);
	U->factor[nrn] = 1.0 / U->factor[nrn];
}

__device__
void nrn_inter_initial(States* S, Parameters* P, Neurons* U, int nrn_seg_index, float V) {
	/**
	initialize channels, based on cropped evaluate_fct function
	*/
	float V_mem = V - V_adj;
	//
	float a = 0.32 * (13.0 - V_mem) / (exp((13.0 - V_mem) / 4.0) - 1.0);
	float b = 0.28 * (V_mem - 40.0) / (exp((V_mem - 40.0) / 5.0) - 1.0);
	S->m[nrn_seg_index] = a / (a + b);   // m_inf
	//
	a = 0.128 * exp((17.0 - V_mem) / 18.0);
	b = 4.0 / (1.0 + exp((40.0 - V_mem) / 5.0));
	S->h[nrn_seg_index] = a / (a + b);   // h_inf
	//
	a = 0.032 * (15.0 - V_mem) / (exp((15.0 - V_mem) / 5.0) - 1.0);
	b = 0.5 * exp((10.0 - V_mem) / 40.0);
	S->n[nrn_seg_index] = a / (a + b);   // n_inf
}

__device__
void nrn_moto_initial(States* S, Parameters* P, Neurons* U, int nrn_seg_index, float V) {
	/** initialize channels, based on cropped evaluate_fct function */
	float a = alpham(V);
	S->m[nrn_seg_index] = a / (a + betam(V));                         // m_inf
	S->h[nrn_seg_index] = 1.0 / (1.0 + Exp((V + 65.0) / 7.0));   // h_inf
	S->p[nrn_seg_index] = 1.0 / (1.0 + Exp(-(V + 55.8) / 3.7));  // p_inf
	S->n[nrn_seg_index] = 1.0 / (1.0 + Exp(-(V + 38.0) / 15.0)); // n_inf
	S->mc[nrn_seg_index] = 1.0 / (1.0 + Exp(-(V + 32.0) / 5.0)); // mc_inf
	S->hc[nrn_seg_index] = 1.0 / (1.0 + Exp((V + 50.0) / 5.0));  // hc_inf
	S->cai[nrn_seg_index] = 0.0001;
}

__device__
void nrn_muslce_initial(States* S, Parameters* P, Neurons* U, int nrn_seg_index, float V) {
	/**
	initialize channels, based on cropped evaluate_fct function
	*/
	float V_mem = V - V_adj;
	//
	float a = 0.32 * (13.0 - V_mem) / (exp((13.0 - V_mem) / 4.0) - 1.0);
	float b = 0.28 * (V_mem - 40.0) / (exp((V_mem - 40.0) / 5.0) - 1.0);
	S->m[nrn_seg_index] = a / (a + b);   // m_inf
	//
	a = 0.128 * exp((17.0 - V_mem) / 18.0);
	b = 4.0 / (1.0 + exp((40.0 - V_mem) / 5.0));
	S->h[nrn_seg_index] = a / (a + b);   // h_inf
	//
	a = 0.032 * (15.0 - V_mem) / (exp((15.0 - V_mem) / 5.0) - 1.0);
	b = 0.5 * exp((10.0 - V_mem) / 40.0);
	S->n[nrn_seg_index] = a / (a + b);   // n_inf
}

__device__
void recalc_inter_channels(States* S, Parameters* P, Neurons* U, int nrn_seg_index, float V) {
	/** calculate new states of channels (evaluate_fct) */
	// BREAKPOINT -> states -> evaluate_fct
	float V_mem = V - V_adj;
	//
	float a = 0.32 * (13.0 - V_mem) / (exp((13.0 - V_mem) / 4.0) - 1.0);
	float b = 0.28 * (V_mem - 40.0) / (exp((V_mem - 40.0) / 5.0) - 1.0);
	float tau = 1.0 / (a + b);
	float inf = a / (a + b);
	S->m[nrn_seg_index] += (1.0 - exp(-dt / tau)) * (inf - S->m[nrn_seg_index]);
	//
	a = 0.128 * exp((17.0 - V_mem) / 18.0);
	b = 4.0 / (1.0 + exp((40.0 - V_mem) / 5.0));
	tau = 1.0 / (a + b);
	inf = a / (a + b);
	S->h[nrn_seg_index] += (1.0 - exp(-dt / tau)) * (inf - S->h[nrn_seg_index]);
	//
	a = 0.032 * (15.0 - V_mem) / (exp((15.0 - V_mem) / 5.0) - 1.0);
	b = 0.5 * exp((10.0 - V_mem) / 40.0);
	tau = 1.0 / (a + b);
	inf = a / (a + b);
	// states
	S->n[nrn_seg_index] += (1.0 - exp(-dt / tau)) * (inf - S->n[nrn_seg_index]);
}

__device__
void recalc_moto_channels(States* S, Parameters* P, Neurons* U, int nrn_seg_index, float V) {
	/** calculate new states of channels (evaluate_fct) */
	//  BREAKPOINT -> states -> evaluate_fct
	float a = alpham(V);
	float b = betam(V);
	// m
	float tau = 1.0 / (a + b);
	float inf = a / (a + b);
	S->m[nrn_seg_index] += (1.0 - exp(-dt / tau)) * (inf - S->m[nrn_seg_index]);
	// h
	tau = 30.0 / (Exp((V + 60.0) / 15.0) + Exp(-(V + 60.0) / 16.0));
	inf = 1.0 / (1.0 + Exp((V + 65.0) / 7.0));
	S->h[nrn_seg_index] += (1.0 - exp(-dt / tau)) * (inf - S->h[nrn_seg_index]);
	// DELAYED RECTIFIER POTASSIUM
	tau = 5.0 / (Exp((V + 50.0) / 40.0) + Exp(-(V + 50.0) / 50.0));
	inf = 1.0 / (1.0 + Exp(-(V + 38.0) / 15.0));
	S->n[nrn_seg_index] += (1.0 - exp(-dt / tau)) * (inf - S->n[nrn_seg_index]);
	// CALCIUM DYNAMICS L-type
	tau = 400.0;
	inf = 1.0 / (1.0 + Exp(-(V + 55.8) / 3.7));
	S->p[nrn_seg_index] += (1.0 - exp(-dt / tau)) * (inf - S->p[nrn_seg_index]);
	// CALCIUM DYNAMICS N-type
	float mc_inf = 1.0 / (1.0 + Exp(-(V + 32.0) / 5.0));
	float hc_inf = 1.0 / (1.0 + Exp((V + 50.0) / 5.0));
	S->mc[nrn_seg_index] += (1.0 - exp(-dt / 15.0)) * (mc_inf - S->mc[nrn_seg_index]);    // tau_mc = 15
	S->hc[nrn_seg_index] += (1.0 - exp(-dt / 50.0)) * (hc_inf - S->hc[nrn_seg_index]);    // tau_hc = 50
	S->cai[nrn_seg_index] += (1.0 - exp(-dt * 0.04)) * (-0.01 * S->I_Ca[nrn_seg_index] / 0.04 - S->cai[nrn_seg_index]);
}

__device__
void recalc_muslce_channels(States* S, Parameters* P, Neurons* U, int nrn_seg_index, float V) {
	/** calculate new states of channels (evaluate_fct) */
	// BREAKPOINT -> states -> evaluate_fct
	float V_mem = V - V_adj;
	//
	float a = 0.32 * (13.0 - V_mem) / (exp((13.0 - V_mem) / 4.0) - 1.0);
	float b = 0.28 * (V_mem - 40.0) / (exp((V_mem - 40.0) / 5.0) - 1.0);
	float tau = 1.0 / (a + b);
	float inf = a / (a + b);
	S->m[nrn_seg_index] += (1.0 - exp(-dt / tau)) * (inf - S->m[nrn_seg_index]);
	//
	a = 0.128 * exp((17.0 - V_mem) / 18.0);
	b = 4.0 / (1.0 + exp((40.0 - V_mem) / 5.0));
	tau = 1.0 / (a + b);
	inf = a / (a + b);
	S->h[nrn_seg_index] += (1.0 - exp(-dt / tau)) * (inf - S->h[nrn_seg_index]);
	//
	a = 0.032 * (15.0 - V_mem) / (exp((15.0 - V_mem) / 5.0) - 1.0);
	b = 0.5 * exp((10.0 - V_mem) / 40.0);
	tau = 1.0 / (a + b);
	inf = a / (a + b);
	S->n[nrn_seg_index] += (1.0 - exp(-dt / tau)) * (inf - S->n[nrn_seg_index]);
	//
	float qt = pow(q10, (celsius - 33.0) / 10.0);
	float linf = 1.0 / (1.0 + exp((V - vhalfl) / kl)); // l_steadystate
	float taul = 1.0 / (qt * (at * exp(-V / vhalft) + bt * exp(V / vhalft)));
	float alpha = 0.3 / (1.0 + exp((V + 43.0) / -5.0));
	float beta = 0.03 / (1.0 + exp((V + 80.0) / -1.0));
	float summ = alpha + beta;
	float stau = 1.0 / summ;
	float sinf = alpha / summ;
	// states
	S->l[nrn_seg_index] += (1.0 - exp(-dt / taul)) * (linf - S->l[nrn_seg_index]);
	S->s[nrn_seg_index] += (1.0 - exp(-dt / stau)) * (sinf - S->s[nrn_seg_index]);
}

__device__
void nrn_rhs_ext(int nrn) {

}

__device__
void nrn_setup_ext(int nrn) {

}

__device__
void nrn_update_2d(int nrn) {

}

__device__
void nrn_rhs(States* S, Parameters* P, Neurons* U, int nrn, int i1, int i3) {
	/**
	void nrn_rhs(NrnThread *_nt) combined with the first part of nrn_lhs
	calculate right hand side of
	cm*dvm/dt = -i(vm) + is(vi) + ai_j*(vi_j - vi)
	cx*dvx/dt - cm*dvm/dt = -gx*(vx - ex) + i(vm) + ax_j*(vx_j - vx)
	This is a common operation for fixed step, cvode, and daspk methods
	*/
	// init _rhs and _lhs (NODE_D) as zero
	for (int i = i1; i < i3; ++i) {
		S->NODE_RHS[i] = 0.0;
		S->NODE_D[i] = 0.0;
//		ext_rhs[i1:i3, :] = 0
	}

	// update MOD rhs, CAPS has no current [CAP MOD CAP]!
	int center_segment = i1 + ((P->models[nrn] == MUSCLE)? 2 : 1);
	// update segments except CAPs
	float V, _g, _rhs;
	for (int nrn_seg = i1 + 1; nrn_seg < i3 - 1; ++nrn_seg) {
		V = S->Vm[nrn_seg];
		// SYNAPTIC update
		if (nrn_seg == center_segment) {
			// static void nrn_cur(_NrnThread* _nt, _Memb_list* _ml, int _type)
			_g = syn_current(U, P, nrn, V + 0.001);
			_rhs = syn_current(U, P, nrn, V);
			_g = (_g - _rhs) / .001;
			_g *= 1.e2 / S->NODE_AREA[nrn_seg];
			_rhs *= 1.e2 / S->NODE_AREA[nrn_seg];
			S->NODE_RHS[nrn_seg] -= _rhs;
			// static void nrn_jacob(_NrnThread* _nt, _Memb_list* _ml, int _type)
			S->NODE_D[nrn_seg] += _g;
		}
		// NEURON update
		// static void nrn_cur(_NrnThread* _nt, _Memb_list* _ml, int _type)
		if (P->models[nrn] == INTER) {
			// muscle and inter has the same fast_channel function
			_g = nrn_fastchannel_current(S, P, U, nrn, nrn_seg, V + 0.001);
			_rhs = nrn_fastchannel_current(S, P, U, nrn, nrn_seg, V);
		} else if (P->models[nrn] == MOTO) {
			_g = nrn_moto_current(S, P, U, nrn, nrn_seg, V + 0.001);
			_rhs = nrn_moto_current(S, P, U, nrn, nrn_seg, V);
		} else if (P->models[nrn] == MUSCLE) {
			// muscle and inter has the same fast_channel function
			_g = nrn_fastchannel_current(S, P, U, nrn, nrn_seg, V + 0.001);
			_rhs = nrn_fastchannel_current(S, P, U, nrn, nrn_seg, V);
		} else {
			// todo
		}

		// save data like in NEURON (after .mod nrn_cur)
		_g = (_g - _rhs) / 0.001;
		S->NODE_RHS[nrn_seg] -= _rhs;
		// static void nrn_jacob(_NrnThread* _nt, _Memb_list* _ml, int _type)
		S->NODE_D[nrn_seg] += _g;
		// end FOR segments
	}
	// activsynapse_rhs()
	if (EXTRACELLULAR) {
		// Cannot have any axial terms yet so that i(vm) can be calculated from
		// i(vm)+is(vi) and is(vi) which are stored in rhs vector.
		nrn_rhs_ext(nrn);
		// nrn_rhs_ext has also computed the the internal axial current for those
		// nodes containing the extracellular mechanism
	}
	// activstim_rhs()
	// activclamp_rhs()

	// todo: always 0, because Vm0 = Vm1 = Vm2 at [CAP node CAP] model (1 section)
	float dv;
	for (int nrn_seg = i1 + 1; nrn_seg < i3; ++nrn_seg) {
		dv = S->Vm[nrn_seg - 1] - S->Vm[nrn_seg];
		// our connection coefficients are negative so
		S->NODE_RHS[nrn_seg] -= S->NODE_B[nrn_seg] * dv;
		S->NODE_RHS[nrn_seg - 1] += S->NODE_A[nrn_seg] * dv;
	}
}

__device__
void bksub(States* S, Parameters* P, Neurons* U, int nrn, int i1, int i3) {
	/**
	void bksub(NrnThread* _nt)
	*/
	// intracellular
	// note that loop from i1 to i1 + 1 is always SINGLE element
	S->NODE_RHS[i1] /= S->NODE_D[i1];
	//
	for (int nrn_seg = i1 + 1; nrn_seg < i3; ++nrn_seg) {
		S->NODE_RHS[nrn_seg] -= S->NODE_B[nrn_seg] * S->NODE_RHS[nrn_seg - 1];
		S->NODE_RHS[nrn_seg] /= S->NODE_D[nrn_seg];
	}
	// extracellular
	if (EXTRACELLULAR) {
//		for j in range(nlayer):
//	ext_rhs[i1, j] /= ext_d[i1, j]
//	for nrn_seg in range(i1 + 1, i3):
//	for j in range(nlayer):
//	ext_rhs[nrn_seg, j] -= ext_b[nrn_seg, j] * ext_rhs[nrn_seg - 1, j]
//	ext_rhs[nrn_seg, j] /= ext_d[nrn_seg, j]
	}
}

__device__
void triang(States* S, Parameters* P, Neurons* U, int nrn, int i1, int i3) {
	/**
	void triang(NrnThread* _nt)
	*/
	// intracellular
	float ppp;
	int nrn_seg = i3 - 1;
	while (nrn_seg >= i1 + 1) {
		ppp = S->NODE_A[nrn_seg] / S->NODE_D[nrn_seg];
		S->NODE_D[nrn_seg - 1] -= ppp * S->NODE_B[nrn_seg];
		S->NODE_RHS[nrn_seg - 1] -= ppp * S->NODE_RHS[nrn_seg];
		nrn_seg--;
	}
	// extracellular
	if (EXTRACELLULAR) {
//		nrn_seg = i3 - 1
//		while nrn_seg >= i1 + 1:
//			for j in range(nlayer):
//				ppp = ext_a[nrn_seg, j] / ext_d[nrn_seg, j]
//				ext_d[nrn_seg - 1, j] -= ppp * ext_b[nrn_seg, j]
//				ext_rhs[nrn_seg - 1, j] -= ppp * ext_rhs[nrn_seg, j]
//			nrn_seg--
	}
}

__device__
void nrn_solve(States* S, Parameters* P, Neurons* U, int nrn, int i1, int i3) {
	/**
	void nrn_solve(NrnThread* _nt)
	*/
	triang(S, P, U, nrn, i1, i3);
	bksub(S, P, U, nrn, i1, i3);
}

__device__
void setup_tree_matrix(States* S, Parameters* P, Neurons* U, int nrn, int i1, int i3) {
	/** void setup_tree_matrix(NrnThread* _nt) */
	nrn_rhs(S, P, U, nrn, i1, i3);
	// simplified nrn_lhs(nrn)
	for (int i = i1; i < i3; ++i) {
		S->NODE_D[i] += S->const_NODE_D[i];
	}
}

__device__
void update(States* S, Parameters* P, Neurons* U, int nrn, int i1, int i3) {
	/** void update(NrnThread* _nt) */
	// final voltage updating
	for (int nrn_seg = i1; nrn_seg < i3; ++nrn_seg) {
		S->Vm[nrn_seg] += S->NODE_RHS[nrn_seg];
	}
	// save data like in NEURON (after .mod nrn_cur)
//	if DEBUG and nrn in save_neuron_ids:
//	save_data()
	// extracellular
	nrn_update_2d(nrn);
}

__device__
void nrn_deliver_events(States* S, Parameters* P, Neurons* U, int nrn) {
/** void nrn_deliver_events(NrnThread* nt) */
	// get the central segment (for detecting spikes): i1 + (2 or 1)
	int seg_update = P->nrn_start_seg[nrn] + ((P->models[nrn] == MUSCLE)? 2 : 1);
	// check if neuron has spike with special flag for avoidance multi-spike detecting
	if (!U->spike_on[nrn] && S->Vm[seg_update] > V_th) {
		U->spike_on[nrn] = true;
		U->has_spike[nrn] = true;
	} else if (S->Vm[seg_update] < V_th) {
		U->spike_on[nrn] = false;
	}
}

__device__
void nrn_fixed_step_lastpart(States* S, Parameters* P, Neurons* U, int nrn, int i1, int i3) {
	/**
	void *nrn_fixed_step_lastpart(NrnThread *nth)
	*/
	// update neurons' synapses state
	recalc_synaptic(S, P, U, nrn);
	//  update neurons' segments state
	if (P->models[nrn] == INTER) {
		for(int nrn_seg = i1; nrn_seg < i3; ++nrn_seg) {
			recalc_inter_channels(S, P, U, nrn_seg, S->Vm[nrn_seg]);
		}
	} else if (P->models[nrn] == INTER) {
		for(int nrn_seg = i1; nrn_seg < i3; ++nrn_seg) {
			recalc_moto_channels(S, P, U, nrn_seg, S->Vm[nrn_seg]);
		}
	} else if (P->models[nrn] == MUSCLE) {
		for(int nrn_seg = i1; nrn_seg < i3; ++nrn_seg) {
			recalc_muslce_channels(S, P, U, nrn_seg, S->Vm[nrn_seg]);
		}
	} else {

	}
	//  spike detection for (in synapse kernel)
	nrn_deliver_events(S, P, U, nrn);
}

__device__
void nrn_area_ri(States* S, Parameters* P, Neurons* U) {
	/**
	void nrn_area_ri(Section *sec) [790] treeset.c
	area for right circular cylinders. Ri as right half of parent + left half of this
	*/
	for (int nrn = 0; nrn < U->size; ++nrn) {
		if (P->models[nrn] == GENERATOR)
			continue;
		int i1 = P->nrn_start_seg[nrn];
		int i3 = P->nrn_start_seg[nrn + 1];
		int nrn_seg, segments = (i3 - i1 - 2);
		// dx = section_length(sec) / ((double) (sec->nnode - 1));
		float dx = P->length[nrn] / segments; // divide by the last index of node (or segments count)
		float rright = 0, rleft;
		// todo sec->pnode needs +1 index
		for (nrn_seg = i1 + 1; nrn_seg < i1 + segments + 1; ++nrn_seg) {
			// area for right circular cylinders. Ri as right half of parent + left half of this
			S->NODE_AREA[nrn_seg] = PI * dx * P->diam[nrn];
			rleft = 1e-2 * P->Ra[nrn] * (dx / 2.0) / (PI * pow(P->diam[nrn], 2) / 4.0);   // left half segment Megohms
			S->NODE_RINV[nrn_seg] = 1.0 / (rleft + rright); // uS
			rright = rleft;
		}
		//the first and last segments has zero length. Area is 1e2 in dimensionless units
		S->NODE_AREA[i1] = 100.0;
		nrn_seg = i1 + segments + 1; // the last segment
		S->NODE_AREA[nrn_seg] = 100.0;
		S->NODE_RINV[nrn_seg] = 1.0 / rright;
	}
}

__device__
void ext_con_coef(States* S, Parameters* P, Neurons* U) {

}

__device__
void connection_coef(States* S, Parameters* P, Neurons* U) {
	/** void connection_coef(void) treeset.c */
	nrn_area_ri(S, P, U);
	// NODE_A is the effect of this node on the parent node's equation
	// NODE_B is the effect of the parent node on this node's equation
	for (int nrn =0; nrn < U->size; ++nrn) {
		if (P->models[nrn] == GENERATOR)
			continue;
		int i1 = P->nrn_start_seg[nrn];
		int i3 = P->nrn_start_seg[nrn + 1];
		int segments = (i3 - i1 - 2);
		// first the effect of node on parent equation. Note that last nodes have area = 1.e2 in dimensionless
		// units so that last nodes have units of microsiemens
		// todo sec->pnode needs +1 index
		int nrn_seg = i1 + 1;
		// sec->prop->dparam[4].val = 1, what is dparam[4].val
		S->NODE_A[nrn_seg] = -1.e2 * 1.0 * S->NODE_RINV[nrn_seg] / S->NODE_AREA[nrn_seg - 1];
		// todo sec->pnode needs +1 index
		for (nrn_seg = i1 + 1 + 1; nrn_seg < i1 + segments + 1 + 1; ++nrn_seg) {
			S->NODE_A[nrn_seg] = -1.e2 * S->NODE_RINV[nrn_seg] / S->NODE_AREA[nrn_seg - 1];
		}
		// now the effect of parent on node equation
		// todo sec->pnode needs +1 index
		for (nrn_seg = i1 + 1; nrn_seg < i1 + segments + 1 + 1; ++nrn_seg) {
			S->NODE_B[nrn_seg] = -1.e2 * S->NODE_RINV[nrn_seg] / S->NODE_AREA[nrn_seg];
		}
	}
	// for extracellular
	ext_con_coef(S, P, U);
	/// note: from LHS, this functions just recalc each time the constant NODED (!)
	/**
	void nrn_lhs(NrnThread *_nt)
	NODE_D[nrn, nd] updating is located at nrn_rhs, because _g is not the global variable
	*/
	// nt->cj = 2/dt if (secondorder) else 1/dt
	// note, the first is CAP
	// function nrn_cap_jacob(_nt, _nt->tml->ml);
	float cj = 1.0 / dt;
	float cfac = 0.001 * cj;
	for (int nrn = 0; nrn < U->size; ++nrn) {
		if (P->models[nrn] == GENERATOR)
			continue;
		int i1 = P->nrn_start_seg[nrn];
		int i3 = P->nrn_start_seg[nrn + 1];
		int segments = (i3 - i1 - 2);
		for (int nrn_seg = i1 + 1; nrn_seg < i1 + segments + 1; ++nrn_seg) {  // added + 1 for nodelist
			S->const_NODE_D[nrn_seg] += cfac * P->Cm[nrn];
		}
		// updating NODED
		for (int nrn_seg = i1 + 1; nrn_seg < i3; ++nrn_seg) {
			S->const_NODE_D[nrn_seg] -= S->NODE_B[nrn_seg];
			S->const_NODE_D[nrn_seg - 1] -= S->NODE_A[nrn_seg];
		}
	}
	// extra
	// _a_matelm += NODE_A[nrn, nd]
	// _b_matelm += NODE_B[nrn, nd]
}

__global__
void initialize_kernel(States* S, Parameters* P, Neurons* U, float v_init) {
	/** */
	// todo do not invoke for generators
	connection_coef(S, P, U);
	// for different models -- different init function
	for (int nrn = 0; nrn < U->size; ++nrn) {
		// do not init neuron state for generator
		if (P->models[nrn] == GENERATOR)
			continue;
		int i1 = P->nrn_start_seg[nrn];
		int i3 = P->nrn_start_seg[nrn + 1];
		// for each segment init the neuron model
		for (int nrn_seg = i1; nrn_seg < i3; ++nrn_seg) {
			S->Vm[nrn_seg] = v_init;
			if (P->models[nrn] == INTER) {
				nrn_inter_initial(S, P, U, nrn_seg, v_init);
			} else if (P->models[nrn] == MOTO) {
				nrn_moto_initial(S, P, U, nrn_seg, v_init);
			} else if (P->models[nrn] == MUSCLE) {
				nrn_muslce_initial(S, P, U, nrn_seg, v_init);
			} else {

			}
		}
		// init RHS/LHS
		setup_tree_matrix(S, P, U, nrn, i1, i3);
		// init tau synapses
		syn_initial(S, P, U, nrn);
	}
}

__global__
void neuron_kernel(States *S, Parameters *P, Neurons *U, int t) {
	/// STRIDE neuron update
	for (int nrn = blockIdx.x * blockDim.x + threadIdx.x; nrn < U->size; nrn += blockDim.x * gridDim.x) {
		//
		U->has_spike[nrn] = false;
		// generator
		if (P->models[nrn] == GENERATOR) {
			// EES
			if (nrn == 0 && (t % (int)(1000 / 40 / dt) == 0)) {
				U->has_spike[nrn] = true;
			}
		} else {
			int i1 = P->nrn_start_seg[nrn];
			int i3 = P->nrn_start_seg[nrn + 1];
			setup_tree_matrix(S, P, U, nrn, i1, i3);
			nrn_solve(S, P, U, nrn, i1, i3);
			update(S, P, U, nrn, i1, i3);
			nrn_fixed_step_lastpart(S, P, U, nrn, i1, i3);
		}
	}
}

__global__
void synapse_kernel(Neurons *U, Synapses* synapses) {
	/**
	void deliver_net_events(NrnThread* nt)
	*/
	int pre_nrn, post_id;
	float weight;
	for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < synapses->size; index += blockDim.x * gridDim.x) {
		pre_nrn = synapses->syn_pre_nrn[index];
		if (U->has_spike[pre_nrn] && synapses->syn_delay_timer[index] == -1) {
			synapses->syn_delay_timer[index] = synapses->syn_delay[index];
		}
		if (synapses->syn_delay_timer[index] == 0) {
			post_id = synapses->syn_post_nrn[index];
			weight = synapses->syn_weight[index];
			if (weight >= 0) {
				atomicAdd(&U->g_exc[post_id], weight);
			} else {
				atomicAdd(&U->g_inh_A[post_id], -weight * U->factor[post_id]);
				atomicAdd(&U->g_inh_B[post_id], -weight * U->factor[post_id]);
				synapses->syn_delay_timer[index] = -1;
			}
		}
		if (synapses->syn_delay_timer[index] > 0) {
			synapses->syn_delay_timer[index]--;
		}
	}
}

void conn_a2a(const Group &pre_neurons, const Group &post_neurons, float delay, float weight) {
	/** */
	for (int pre = pre_neurons.id_start; pre <= pre_neurons.id_end; ++pre) {
		for (int post = post_neurons.id_start; post <= post_neurons.id_end; ++post) {
			// weight = random.gauss(weight, weight / 5)
			// delay = random.gauss(delay, delay / 5)
			vector_syn_pre_nrn.push_back(pre);
			vector_syn_post_nrn.push_back(post);
			vector_syn_weight.push_back(weight);
			vector_syn_delay.push_back((int) (delay / dt));
			vector_syn_delay_timer.push_back(-1);
		}
	}
//	printf("Connect %s to %s [fixed_outdegree] (1:%d). Total: %d W=%.2f, D=%.1f\n",
//	       pre_neurons.group_name.c_str(), post_neurons.group_name.c_str(),
//	       outdegree, pre_neurons.group_size * outdegree, syn_weight, syn_delay);
}

void connect_fixed_outdegree(const Group &pre_neurons,
                             const Group &post_neurons,
                             float delay,
                             float weight,
                             int indegree = 50) {
	// pre_nrns_ids = pre_group[1]
	// post_nrns_ids = post_group[1]
	uniform_int_distribution<int> nsyn_distr(indegree - 15, indegree);
	uniform_int_distribution<int> pre_nrns_ids(pre_neurons.id_start, pre_neurons.id_end);

	int pre, nsyn = nsyn_distr(generator);
	// nsyn = random.randint(indegree - 15, indegree)
	for (int post = post_neurons.id_start; post <= post_neurons.id_end; ++post) {
		for (int _ = 0; _ < nsyn; ++_) {
			pre = pre_nrns_ids(generator);
			// weight = random.gauss(weight, weight / 5)
			// delay = random.gauss(delay, delay / 5)
			vector_syn_pre_nrn.push_back(pre);
			vector_syn_post_nrn.push_back(post);
			vector_syn_weight.push_back(weight);
			vector_syn_delay.push_back((int)(delay / dt));
			vector_syn_delay_timer.push_back(-1);
		}
	}

//	printf("Connect %s to %s [fixed_outdegree] (1:%d). Total: %d W=%.2f, D=%.1f\n",
//	       pre_neurons.group_name.c_str(), post_neurons.group_name.c_str(),
//	       outdegree, pre_neurons.group_size * outdegree, syn_weight, syn_delay);
}


void file_writing(int test_index, GroupMetadata &metadata, const string &folder) {
	ofstream file;
	string file_name = "/dat/" + to_string(test_index) + "_" + metadata.group.group_name + ".dat";

	file.open(folder + file_name);
	// save voltage
	for (unsigned int sim_iter = 0; sim_iter < SIM_TIME_IN_STEPS; sim_iter++)
		file << metadata.voltage_array[sim_iter] << " ";
	file << endl;

	// save g_exc
	for (unsigned int sim_iter = 0; sim_iter < SIM_TIME_IN_STEPS; sim_iter++)
		file << metadata.g_exc[sim_iter] << " ";
	file << endl;

	// save g_inh
	for (unsigned int sim_iter = 0; sim_iter < SIM_TIME_IN_STEPS; sim_iter++)
		file << metadata.g_inh[sim_iter] << " ";
	file << endl;

	// save spikes
	for (float const &value: metadata.spike_vector) {
		file << value << " ";
	}
	file.close();

	cout << "Saved to: " << folder + file_name << endl;
}

void save(vector<Group> groups) {
	for (Group &group : groups) {
		GroupMetadata new_meta(group, SIM_TIME_IN_STEPS);
		saving_groups.emplace_back(new_meta);
	}
}

void copy_data_to(GroupMetadata& metadata,
                  const float* Vm,
                  const float* g_exc,
                  const float* g_inh_A,
                  const float* g_inh_B,
                  const bool* has_spike,
                  const unsigned int sim_iter) {
	float nrn_mean_volt = 0;
	float nrn_mean_g_exc = 0;
	float nrn_mean_g_inh = 0;

	int center;
	for (unsigned int nrn = metadata.group.id_start; nrn <= metadata.group.id_end; ++nrn) {
		center = vector_nrn_start_seg[nrn] + ((vector_models[nrn] == MUSCLE)? 2 : 1);
		nrn_mean_volt += Vm[center];
		nrn_mean_g_exc += g_exc[nrn];
		nrn_mean_g_inh += (g_inh_B[nrn] - g_inh_A[nrn]);
		if (has_spike[nrn]) {
			metadata.spike_vector.push_back(step_to_ms(sim_iter));
		}
	}
	metadata.voltage_array[sim_iter] = nrn_mean_volt / metadata.group.group_size;
	metadata.g_exc[sim_iter] = nrn_mean_g_exc / metadata.group.group_size;
	metadata.g_inh[sim_iter] = nrn_mean_g_inh / metadata.group.group_size;
}


void save_result(int test_index) {
	string current_path = getcwd(nullptr, 0);

	printf("[Test #%d] Save results to: %s \n", test_index, current_path.c_str());

	for (GroupMetadata &metadata : saving_groups) {
		file_writing(test_index, metadata, current_path);
	}
}

void init_network() {
	Group gen = form_group("gen", 1, GENERATOR, 1);
	Group OM1 = form_group("OM1", 50, INTER, 1);
	Group OM2 = form_group("OM2", 50, INTER, 1);
	Group OM3 = form_group("OM3", 50, INTER, 1);
	Group moto = form_group("moto", 50, MOTO, 1);
	Group muscle = form_group("muscle", 1, MUSCLE, 3);

	conn_a2a(gen, OM1, 1, 1.5);

	connect_fixed_outdegree(OM1, OM2, 2, 1.85);
	connect_fixed_outdegree(OM2, OM1, 3, 1.85);
	connect_fixed_outdegree(OM2, OM3, 3, 0.00055);
	connect_fixed_outdegree(OM1, OM3, 3, 0.00005);
	connect_fixed_outdegree(OM3, OM2, 1, -4.5);
	connect_fixed_outdegree(OM3, OM1, 1, -4.5);
	connect_fixed_outdegree(OM2, moto, 2, 1.5);
	connect_fixed_outdegree(moto, muscle, 2, 15.5);

	vector<Group> groups = {OM1, OM2, OM3, moto, muscle};
	save(groups);

	vector_nrn_start_seg.push_back(nrns_and_segs);
}

void simulate(int test_index) {
	/**
	 *
	 */
	// init structs
	States *S = (States *)malloc(sizeof(States));
	Parameters *P = (Parameters *)malloc(sizeof(Parameters));
	Neurons *U = (Neurons *)malloc(sizeof(Neurons));
	Synapses *synapses = (Synapses *)malloc(sizeof(Synapses));

	init_network();

	/// GPU
	// init States CPU arrays
	auto *Vm = new float[nrns_and_segs]();
	auto *n = new float[nrns_and_segs]();
	auto *m = new float[nrns_and_segs]();
	auto *h = new float[nrns_and_segs]();
	auto *l = new float[nrns_and_segs]();
	auto *s = new float[nrns_and_segs]();
	auto *p = new float[nrns_and_segs]();
	auto *hc = new float[nrns_and_segs]();
	auto *mc = new float[nrns_and_segs]();
	auto *cai = new float[nrns_and_segs]();
	auto *I_Ca = new float[nrns_and_segs]();
	auto *NODE_A = new float[nrns_and_segs]();
	auto *NODE_B = new float[nrns_and_segs]();
	auto *NODE_D = new float[nrns_and_segs]();
	auto *const_NODE_D = new float[nrns_and_segs]();
	auto *NODE_RHS = new float[nrns_and_segs]();
	auto *NODE_RINV = new float[nrns_and_segs]();
	auto *NODE_AREA = new float[nrns_and_segs]();
	//
	auto *has_spike = new bool[nrns_number]();
	auto *spike_on = new bool[nrns_number]();
	auto *g_exc = new float[nrns_number]();
	auto *g_inh_A = new float[nrns_number]();
	auto *g_inh_B = new float[nrns_number]();
	auto *factor = new float[nrns_number]();


	/// GPU
	// init Parameters (malloc + memcpy) GPU arrays based on CPU vectors
	short *gpu_nrn_start_seg = init_gpu_arr(vector_nrn_start_seg);
	char *gpu_models = init_gpu_arr(vector_models);
	auto *gpu_Cm = init_gpu_arr(vector_Cm);
	float *gpu_gnabar = init_gpu_arr(vector_gnabar);
	float *gpu_gkbar = init_gpu_arr(vector_gkbar);
	float *gpu_gl = init_gpu_arr(vector_gl);
	float *gpu_Ra = init_gpu_arr(vector_Ra);
	float *gpu_diam = init_gpu_arr(vector_diam);
	float *gpu_length = init_gpu_arr(vector_length);
	float *gpu_ena = init_gpu_arr(vector_ena);
	float *gpu_ek = init_gpu_arr(vector_ek);
	float *gpu_el = init_gpu_arr(vector_el);
	float *gpu_gkrect = init_gpu_arr(vector_gkrect);
	float *gpu_gcaN = init_gpu_arr(vector_gcaN);
	float *gpu_gcaL = init_gpu_arr(vector_gcaL);
	float *gpu_gcak = init_gpu_arr(vector_gcak);
	float *gpu_E_ex = init_gpu_arr(vector_E_ex);
	float *gpu_E_inh = init_gpu_arr(vector_E_inh);
	float *gpu_tau_exc = init_gpu_arr(vector_tau_exc);
	float *gpu_tau_inh1 = init_gpu_arr(vector_tau_inh1);
	float *gpu_tau_inh2 = init_gpu_arr(vector_tau_inh2);
	// init Synapses
	int* gpu_syn_pre_nrn = init_gpu_arr(vector_syn_pre_nrn);
	int* gpu_syn_post_nrn = init_gpu_arr(vector_syn_post_nrn);
	float* gpu_syn_weight = init_gpu_arr(vector_syn_weight);
	int* gpu_syn_delay = init_gpu_arr(vector_syn_delay);
	int* gpu_syn_delay_timer = init_gpu_arr(vector_syn_delay_timer);

	// init States GPU arrays based on CPU arrays
	auto *gpu_Vm = init_gpu_arr(Vm, nrns_and_segs);
	auto *gpu_n = init_gpu_arr(n, nrns_and_segs);
	auto *gpu_m = init_gpu_arr(m, nrns_and_segs);
	auto *gpu_h = init_gpu_arr(h, nrns_and_segs);
	auto *gpu_l = init_gpu_arr(l, nrns_and_segs);
	auto *gpu_s = init_gpu_arr(s, nrns_and_segs);
	auto *gpu_p = init_gpu_arr(p, nrns_and_segs);
	auto *gpu_hc = init_gpu_arr(hc, nrns_and_segs);
	auto *gpu_mc = init_gpu_arr(mc, nrns_and_segs);
	auto *gpu_cai = init_gpu_arr(cai, nrns_and_segs);
	auto *gpu_I_Ca = init_gpu_arr(I_Ca, nrns_and_segs);
	auto *gpu_NODE_A = init_gpu_arr(NODE_A, nrns_and_segs);
	auto *gpu_NODE_B = init_gpu_arr(NODE_B, nrns_and_segs);
	auto *gpu_NODE_D = init_gpu_arr(NODE_D, nrns_and_segs);
	auto *gpu_const_NODE_D = init_gpu_arr(const_NODE_D, nrns_and_segs);
	auto *gpu_NODE_RHS = init_gpu_arr(NODE_RHS, nrns_and_segs);
	auto *gpu_NODE_RINV = init_gpu_arr(NODE_RINV, nrns_and_segs);
	auto *gpu_NODE_AREA = init_gpu_arr(NODE_AREA, nrns_and_segs);
	//
	auto *gpu_has_spike = init_gpu_arr(has_spike, nrns_number);
	auto *gpu_spike_on = init_gpu_arr(spike_on, nrns_number);
	auto *gpu_g_exc = init_gpu_arr(g_exc, nrns_number);
	auto *gpu_g_inh_A = init_gpu_arr(g_inh_A, nrns_number);
	auto *gpu_g_inh_B = init_gpu_arr(g_inh_B, nrns_number);
	auto *gpu_factor = init_gpu_arr(factor, nrns_number);

	// Point to device pointer in host struct
	// states
	S->Vm = gpu_Vm;
	S->n = gpu_n;
	S->m = gpu_m;
	S->h = gpu_h;
	S->l = gpu_l;
	S->s = gpu_s;
	S->p = gpu_p;
	S->hc = gpu_hc;
	S->mc = gpu_mc;
	S->cai = gpu_cai;
	S->I_Ca = gpu_I_Ca;
	S->NODE_A = gpu_NODE_A;
	S->NODE_B = gpu_NODE_B;
	S->NODE_D = gpu_NODE_D;
	S->const_NODE_D = gpu_const_NODE_D;
	S->NODE_RHS = gpu_NODE_RHS;
	S->NODE_RINV = gpu_NODE_RINV;
	S->NODE_AREA = gpu_NODE_AREA;
	S->size = nrns_and_segs;
	// parameters
	P->nrn_start_seg = gpu_nrn_start_seg;
	P->models = gpu_models;
	P->Cm = gpu_Cm;
	P->gnabar = gpu_gnabar;
	P->gkbar = gpu_gkbar;
	P->gl = gpu_gl;
	P->Ra = gpu_Ra;
	P->diam = gpu_diam;
	P->length = gpu_length;
	P->ena = gpu_ena;
	P->ek = gpu_ek;
	P->el = gpu_el;
	P->gkrect = gpu_gkrect;
	P->gcaN = gpu_gcaN;
	P->gcaL = gpu_gcaL;
	P->gcak = gpu_gcak;
	P->E_ex = gpu_E_ex;
	P->E_inh = gpu_E_inh;
	P->tau_exc = gpu_tau_exc;
	P->tau_inh1 = gpu_tau_inh1;
	P->tau_inh2 = gpu_tau_inh2;
	P->size = nrns_number;
	// Neurons
	U->has_spike = gpu_has_spike;
	U->spike_on = gpu_spike_on;
	U->g_exc = gpu_g_exc;
	U->g_inh_A = gpu_g_inh_A;
	U->g_inh_B = gpu_g_inh_B;
	U->factor = gpu_factor;
	U->size = nrns_number;
	// Synapses
	synapses->syn_pre_nrn = gpu_syn_pre_nrn;
	synapses->syn_post_nrn = gpu_syn_post_nrn;
	synapses->syn_weight = gpu_syn_weight;
	synapses->syn_delay = gpu_syn_delay;
	synapses->syn_delay_timer = gpu_syn_delay_timer;
	synapses->size = vector_syn_delay.size();

	// allocate structs to the device
	auto *dev_S = init_gpu_arr(S, 1);
	auto *dev_P = init_gpu_arr(P, 1);
	auto *dev_U = init_gpu_arr(U, 1);
	auto *dev_synapses = init_gpu_arr(synapses, 1);

	float time;
	hipEvent_t start, stop;
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventRecord(start, 0));

	// call initialisation kernel
	initialize_kernel<<<1, 1>>>(dev_S, dev_P, dev_U, -70.0);

	// the main simulation loop
	for (unsigned int sim_iter = 0; sim_iter < SIM_TIME_IN_STEPS; ++sim_iter) {
		// deliver_net_events, synapse updating and neuron conductance changing
		synapse_kernel<<<5, 256>>>(dev_U, dev_synapses);
		// another neuron's kernel functions
		neuron_kernel<<<10, 32>>>(dev_S, dev_P, dev_U, sim_iter); // block size need to be a multiply of 256

		// copy state data
		HANDLE_ERROR(hipMemcpy(Vm, gpu_Vm, nrns_and_segs * sizeof(*Vm), hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(g_exc, gpu_g_exc, nrns_number * sizeof(*g_exc), hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(g_inh_A, gpu_g_inh_A, nrns_number * sizeof(*g_inh_A), hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(g_inh_B, gpu_g_inh_B, nrns_number * sizeof(*g_inh_B), hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(has_spike, gpu_has_spike, nrns_number * sizeof(*has_spike), hipMemcpyDeviceToHost));

		// fill records arrays
		for (GroupMetadata& metadata : saving_groups) {
			copy_data_to(metadata, Vm, g_exc, g_inh_A, g_inh_B, has_spike, sim_iter);
		}
	}
	HANDLE_ERROR(hipDeviceSynchronize());
	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	HANDLE_ERROR(hipEventElapsedTime(&time, start, stop));

	printf("Elapsed GPU time: %d ms\n", (int) time);
	// clock rate * 10^6 * (interface width / 8) * 2 / 10^9
	float Tbw = 12000 * pow(10, 6) * (128 / 8) * 2 / pow(10, 9);
	printf("Theoretical Bandwidth GPU (2 Ghz, 128 bit): %.2f GB/s\n", Tbw);
	/* Effective bandwidth = ( ( B r + B w ) ÷ 10 9 ) ÷ time
	Here, the effective bandwidth is in units of GB/s, Br is the number of bytes read per kernel, Bw is the number of
	bytes written per kernel, and time is given in seconds.
	For example, to compute the effective bandwidth of a 2048 x 2048 matrix copy, the following formula could be used:
	Effective bandwidth = ( ( 2048 2 × 4 × 2 ) ÷ 10 9 ) ÷ time */
	save_result(test_index);

	HANDLE_ERROR(hipFree(gpu_Vm));
}

int main(int argc, char **argv) {
	int dev = 0;
	hipDeviceProp_t deviceProp;
	HANDLE_ERROR(hipGetDeviceProperties(&deviceProp, dev));
	printf("%s struct of array at ", argv[0]);
	printf("device %d: %s \n", dev, deviceProp.name);
	//
	HANDLE_ERROR(hipSetDevice(dev));
	//
	simulate(0);
	// reset device
	HANDLE_ERROR(hipDeviceReset());
}